#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

typedef unsigned int TColor;

extern "C" hipTextureObject_t texImage;

////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
// Texture object and channel descriptor for image texture
hipTextureObject_t texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

// CUDA array descriptor
hipArray* a_Src;

////////////////////////////////////////////////////////////////////////////////
// Filtering kernels
////////////////////////////////////////////////////////////////////////////////

extern "C" hipError_t CUDA_MallocArray(uchar4 * *h_Src, int imageW, int imageH)
{
	hipError_t error;

	error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
	error = hipMemcpy2DToArray(a_Src, 0, 0, *h_Src, sizeof(uchar4) * imageW, sizeof(uchar4) * imageW, imageH, hipMemcpyHostToDevice);

	hipResourceDesc texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = a_Src;

	hipTextureDesc texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = false;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeWrap;
	texDescr.addressMode[1] = hipAddressModeWrap;
	texDescr.readMode = hipReadModeNormalizedFloat;

	checkCudaErrors(hipCreateTextureObject(&texImage, &texRes, &texDescr, NULL));

	return error;
}

extern "C" hipError_t CUDA_FreeArray() { return hipFreeArray(a_Src); }
