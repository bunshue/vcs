#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Common.h"
#include "BmpUtil.h"

// CUDA kernel to add elements of two arrays
/*
__global__
void addKernel(int n, byte* x, byte* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);

    for (int i = index; i < n; i += stride)
    {
        //�ثe���S���i�ӳo��
        //y[i] = x[i] + y[i];
        y[i] = 0;
        printf(".");
    }
}
*/

__global__ void addKernel(byte* c, const byte* a, const byte* b)
{
    printf("Q");
    int i = threadIdx.x;
    //c[i] = (a[i]/10 + b[i]/10) % 256;
    c[i] = a[i];
    printf("Z");
}

__global__ void vectorAdd(const byte* A, const byte* B, byte* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        //C[i] = (A[i] + B[i]) % 256;
        C[i] = (A[i] / 2 + B[i] / 2) % 256;
    }
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len);

int main(int argc, char** argv)
{
    // initialize CUDA
    findCudaDevice(argc, (const char**)argv);

    int ImgWidth;
    int ImgHeight;
    int ColorDepth;
    ROI ImgSize;
    int res;
    int ImgStride;
    int ImgDataSize;

    char filename_read1[] = "C:\\_git\\vcs\\_1.data\\______test_files1\\ims01.bmp";
    char filename_read2[] = "C:\\_git\\vcs\\_1.data\\______test_files1\\ims03.bmp";

    printf("Ū���ɮ� : %s\n", filename_read1);
    res = PreLoadBmp2(filename_read1, &ImgWidth, &ImgHeight, &ColorDepth);
    if (res != 0)
    {
        printf("\nError: Image file not found or invalid!\n");
        exit(EXIT_FAILURE);
        return 1;
    }
    ImgSize.width = ImgWidth;
    ImgSize.height = ImgHeight;
    ImgDataSize = ImgWidth * (ColorDepth / 8) * ImgHeight;
    printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
    printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
    byte* ImageData1 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
    //printf("ImgStride = %d\n", ImgStride);
    LoadBmpAsData(filename_read1, ImgStride, ImgSize, ImageData1, ColorDepth);

    printf("Ū���ɮ� : %s\n", filename_read2);
    res = PreLoadBmp2(filename_read2, &ImgWidth, &ImgHeight, &ColorDepth);
    if (res != 0)
    {
        printf("\nError: Image file not found or invalid!\n");
        exit(EXIT_FAILURE);
        return 1;
    }
    ImgSize.width = ImgWidth;
    ImgSize.height = ImgHeight;
    printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
    printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
    byte* ImageData2 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
    //printf("ImgStride = %d\n", ImgStride);
    LoadBmpAsData(filename_read2, ImgStride, ImgSize, ImageData2, ColorDepth);

    byte* ImageData3 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
    for (int i = 0; i < ImgWidth * (ColorDepth / 8) * ImgHeight; i++)
    {
        ImageData3[i] = 0x11;
    }

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    byte* d_A = NULL;
    err = hipMalloc((void**)&d_A, ImgDataSize);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    byte* d_B = NULL;
    err = hipMalloc((void**)&d_B, ImgDataSize);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    byte* d_C = NULL;
    err = hipMalloc((void**)&d_C, ImgDataSize);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, ImageData1, ImgDataSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, ImageData2, ImgDataSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int numElements = ImgDataSize / 2;  //�����`�j�p

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    //                   blocks, threads

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");

    //printf("old\n");
    //printData(ImageData1, ImageData2, ImageData3, 20);

    err = hipMemcpy(ImageData3, d_C, ImgDataSize, hipMemcpyDeviceToHost);

    //printf("new\n");
    //printData(ImageData1, ImageData2, ImageData3, 20);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /*
      // Launch kernel on 1M elements on the GPU
      int blockSize = 256;
      int numBlocks = (N + blockSize - 1) / blockSize;

      //addKernel << <numBlocks, blockSize >> > (N, ImageData1, ImageData2);
      //addKernel << <1, size >> > (dev_c, dev_a, dev_b);
      //__global__ void addKernel(byte * c, const byte * a, const byte * b)
      int size = 1;
      addKernel << <1, size >> > (ImageData3, ImageData1, ImageData2);
    */

    // Wait for GPU to finish before accessing on host
    //hipDeviceSynchronize();

    //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
    char filename_write1[] = "x64\\Debug\\ims.new1.bmp";
    char filename_write2[] = "x64\\Debug\\ims.new2.bmp";
    char filename_write3[] = "x64\\Debug\\ims.new3.bmp";
    printf("�s�@�@��bmp�ɮ� : %s\n", filename_write1);
    printf("�s�@�@��bmp�ɮ� : %s\n", filename_write2);
    printf("�s�@�@��bmp�ɮ� : %s\n", filename_write3);
    //ImgStride = 320;

    ImgSize.width = ImgWidth;
    ImgSize.height = ImgHeight;

    ColorDepth = 32;
    DumpBmpData(filename_write1, ImageData1, ImgStride, ImgSize, ColorDepth);
    DumpBmpData(filename_write2, ImageData2, ImgStride, ImgSize, ColorDepth);
    DumpBmpData(filename_write3, ImageData3, ImgStride, ImgSize, ColorDepth);

    //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP

    /*
    // Free memory
    hipFree(ImageData1);
    hipFree(ImageData2);
    */

    printf("ImgWidth = %d\tImgHeight=%d\n", ImgWidth, ImgHeight);
    printf("ColorDepth = %d\n", ColorDepth);
    printf("ImgStride = %d\n", ImgStride);
    printf("DataSize = %d\n", ImgWidth * (ColorDepth / 8) * ImgHeight);

    int DataSize = ImgWidth * (ColorDepth / 8) * ImgHeight;

    /*
    byte* data1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte* data2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte* data3 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    */

    byte* data1;
    byte* data2;
    byte* data3;

    hipMalloc((void**)&data1, DataSize);
    hipMalloc((void**)&data2, DataSize);
    hipMalloc((void**)&data3, DataSize);

    hipMemcpy(data1, ImageData1, DataSize, hipMemcpyHostToDevice);
    hipMemcpy(data2, ImageData2, DataSize, hipMemcpyHostToDevice);

    FreePlane(ImageData1);
    FreePlane(ImageData2);
    FreePlane(ImageData3);


    hipFree(data1);
    hipFree(data2);
    hipFree(data3);


    printf("Test PASSED\n");

    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // finalize
    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_A[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_B[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_C[i]);
    }
    printf("\n");
}
