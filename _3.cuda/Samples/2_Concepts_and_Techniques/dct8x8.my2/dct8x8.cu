#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

// CUDA kernel to add elements of two arrays
/*
__global__
void addKernel(int n, byte* x, byte* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);

    for (int i = index; i < n; i += stride)
    {
        //�ثe���S���i�ӳo��
        //y[i] = x[i] + y[i];
        y[i] = 0;
        printf(".");
    }
}
*/

__global__ void addKernel(byte* c, const byte* a, const byte* b)
{
    printf("Q");
    int i = threadIdx.x;
    //c[i] = (a[i]/10 + b[i]/10) % 256;
    c[i] = a[i];
    printf("Z");
}

__global__ void vecAdd(byte* a, byte* b, byte* c)
{
    int i = threadIdx.x;
    //c[i] = (a[i] + b[i])%256;
    //c[i] = 0;
    c[i] = (a[i] + b[i]) % 256;
}

int main(int argc, char **argv)
{
  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  int ImgWidth;
  int ImgHeight;
  int ColorDepth;
  ROI ImgSize;
  int res;
  int ImgStride;

  char filename_read1[] = "C:\\_git\\vcs\\_1.data\\______test_files1\\ims01.bmp";
  char filename_read2[] = "C:\\_git\\vcs\\_1.data\\______test_files1\\ims03.bmp";

  printf("Ū���ɮ� : %s\n", filename_read1);
  res = PreLoadBmp2(filename_read1, &ImgWidth, &ImgHeight, &ColorDepth);
  if (res != 0)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
  printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
  byte* ImageData1 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  //printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read1, ImgStride, ImgSize, ImageData1, ColorDepth);

  printf("Ū���ɮ� : %s\n", filename_read2);
  res = PreLoadBmp2(filename_read2, &ImgWidth, &ImgHeight, &ColorDepth);
  if (res != 0)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
  printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
  byte* ImageData2 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  //printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read2, ImgStride, ImgSize, ImageData2, ColorDepth);

  byte* ImageData3 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * (ColorDepth / 8) * ImgHeight; i++)
  {
      ImageData3[i] = 0x11;
  }

  int N = 256;
  vecAdd << <1, N >> > (ImageData1, ImageData2, ImageData3);
  //         1 block, N threads

  /*
  byte* ImageData1;
  byte* ImageData2;

  // Allocate Unified Memory - accessible from CPU or GPU
  hipMallocManaged(&ImageData1, N * sizeof(byte));
  hipMallocManaged(&ImageData2, N * sizeof(byte));

  // initialize ImageData1 and ImageData2 arrays on the host
  for (int i = 0; i < N; i++)
  {
      ImageData1[i] = 3.0f;
      ImageData2[i] = 7.0f;
  }
  */

  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }
  printf("\n");

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  //addKernel << <numBlocks, blockSize >> > (N, ImageData1, ImageData2);
  //addKernel << <1, size >> > (dev_c, dev_a, dev_b);
  //__global__ void addKernel(byte * c, const byte * a, const byte * b)
  
  int size = 1;
  addKernel << <1, size >> > (ImageData3, ImageData1, ImageData2);

  for (int i = 0; i < 5; i++)
  {
      //ImageData3[i] = (ImageData1[i] + ImageData2[i]) % 256;


  }

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  printf("\n");
  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write1[] = "x64\\Debug\\ims.new1.bmp";
  char filename_write2[] = "x64\\Debug\\ims.new2.bmp";
  char filename_write3[] = "x64\\Debug\\ims.new3.bmp";

  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write1);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write2);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write3);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  ColorDepth = 32;
  DumpBmpData(filename_write1, ImageData1, ImgStride, ImgSize, ColorDepth);
  DumpBmpData(filename_write2, ImageData2, ImgStride, ImgSize, ColorDepth);
  DumpBmpData(filename_write3, ImageData3, ImgStride, ImgSize, ColorDepth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP

  /*
  // Free memory
  hipFree(ImageData1);
  hipFree(ImageData2);
  */

  printf("ImgWidth = %d\tImgHeight=%d\n", ImgWidth, ImgHeight);
  printf("ColorDepth = %d\n", ColorDepth);
  printf("ImgStride = %d\n", ImgStride);
  printf("DataSize = %d\n", ImgWidth * (ColorDepth / 8) * ImgHeight);

  int DataSize = ImgWidth * (ColorDepth / 8) * ImgHeight;

  /*
  byte* data1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte* data2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte* data3 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  */

  byte* data1;
  byte* data2;
  byte* data3;

  hipMalloc((void**)&data1, DataSize);
  hipMalloc((void**)&data2, DataSize);
  hipMalloc((void**)&data3, DataSize);

  hipMemcpy(data1, ImageData1, DataSize, hipMemcpyHostToDevice);
  hipMemcpy(data2, ImageData2, DataSize, hipMemcpyHostToDevice);

  /*
  for (int i = 0; i < ImgWidth * ImgHeight; i++)
  {
      data1[i] = (i % 256);
      data2[i] = (i % 256);
      data3[i] = 0x17;
  }

  for (int i = 0; i < 10; i++)
  {
      printf("data1[%d] = %d\t", i, data1[i]);
      printf("data2[%d] = %d\t", i, data2[i]);
      printf("data3[%d] = %d\n", i, data3[i]);
  }
  */

  for (int i = 0; i < 10; i++)
  {
      //fail
      //printf("data1[%d] = %d\t", i, *(&data1[0] + i));
      //printf("data2[%d] = %d\t", i, *(&data2[0] + i));
      //printf("data3[%d] = %d\n", i, data3[i]);
  }

  N = 256;
  vecAdd << <1, N >> > (data1, data2, data3);
  //         1 block, N threads

  hipMemcpy(ImageData3, data3, DataSize, hipMemcpyDeviceToHost);

  for (int i = 0; i < 10; i++)
  {
      //fail
      //printf("data1[%d] = %d\t", i, *(data1 + i));
      //printf("data2[%d] = %d\t", i, *(data2 + i));
      //printf("data3[%d] = %d\n", i, *(data3 + i));
  }

  FreePlane(ImageData1);
  FreePlane(ImageData2);
  FreePlane(ImageData3);


  hipFree(data1);
  hipFree(data2);
  hipFree(data3);



  // finalize
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
