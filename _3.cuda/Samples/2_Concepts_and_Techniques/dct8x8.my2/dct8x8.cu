#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
**************************************************************************
* \file dct8x8.cu
* \brief Contains entry point, wrappers to host and device code and benchmark.
*
* This sample implements forward and inverse Discrete Cosine Transform to blocks
* of image pixels (of 8x8 size), as in JPEG standard. The typical work flow is
*as
* follows:
* 1. Run CPU version (Host code) and measure execution time;
* 2. Run CUDA version (Device code) and measure execution time;
* 3. Output execution timings and calculate CUDA speedup.
*/

#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

// CUDA kernel to add elements of two arrays
/*
__global__
void addKernel(int n, byte* x, byte* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);

    for (int i = index; i < n; i += stride)
    {
        //�ثe���S���i�ӳo��
        //y[i] = x[i] + y[i];
        y[i] = 0;
        printf(".");
    }
}
*/

__global__ void addKernel(byte* c, const byte* a, const byte* b)
{
    printf("Q");
    int i = threadIdx.x;
    //c[i] = (a[i]/10 + b[i]/10) % 256;
    c[i] = a[i];
    printf("Z");
}


/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/

int main(int argc, char **argv)
{
  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  char filename1[] = "portrait_noise.bmp";
  char filename2[] = "portrait_noise.ok.bmp";

  // preload image (acquire dimensions)
  int ImgWidth;
  int ImgHeight;
  ROI ImgSize;

  printf("PreLoadBmp, file : %s\n", filename1);
  int res = PreLoadBmp(filename1, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  printf("Ū���ɮ� : %s\n", filename1);

  // check image dimensions are multiples of BLOCK_SIZE
  if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0)
  {
    printf("\nError: Input image dimensions must be multiples of 8!\n");
    exit(EXIT_FAILURE);
    return 1;
  }

  // allocate image buffers
  int ImgStride;
  byte *ImgSrc = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDst = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  // load sample image
  LoadBmpAsGray(filename1, ImgStride, ImgSize, ImgSrc);

  printf("�g�J�ɮ� : %s\n", filename2);
  DumpBmpAsGray(filename2, ImgDst, ImgStride, ImgSize);

  //�s�@�@��24�줸�`�פ�bmp�ɮ� ST
  char filename3[] = "my_bmp333b.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename3);
  ImgStride = 320;

  ImgWidth = 16;
  ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  byte* ImgDst333 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * ImgHeight; i++)
  {
      ImgDst333[i] = (i % 256);
  }
  DumpBmpAsGray(filename3, ImgDst333, ImgStride, ImgSize);
  FreePlane(ImgDst333);
  //�s�@�@��24�줸�`�פ�bmp�ɮ� SP

  // release byte planes
  FreePlane(ImgSrc);
  FreePlane(ImgDst);



  //Ū���@��bmp�ɮ� ST, �P�_�줸�`��
  char filename_read[] = "C:\\______test_files\\pic_256X100b.bmp";
  printf("Ū���ɮ� : %s\n", filename_read);

  res = PreLoadBmp2(filename_read, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);

  int color_depth = GetBmpColorDepth(filename_read);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);


  byte* ImageData = MallocPlaneByte(ImgWidth*(color_depth/8), ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  LoadBmpAsData(filename_read, ImgStride, ImgSize, ImageData, color_depth);

  /*
  for (int i = 0; i < 100; i++)
  {
      printf("%02X ", ImageData[i]);


  }
  printf("\n");
  */

  //���Ʀs���t�@��bmp�ɮ�

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write[] = "pic_256X100b.32.new.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  color_depth = 32;
  DumpBmpData(filename_write, ImageData, ImgStride, ImgSize, color_depth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP


  FreePlane(ImageData);



  char filename_read1[] = "C:\\______test_files\\ims01.bmp";
  char filename_read2[] = "C:\\______test_files\\ims03.bmp";

  printf("Ū���ɮ� : %s\n", filename_read1);
  res = PreLoadBmp2(filename_read1, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  color_depth = GetBmpColorDepth(filename_read1);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);
  byte* ImageData1 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read1, ImgStride, ImgSize, ImageData1, color_depth);

  printf("Ū���ɮ� : %s\n", filename_read2);
  res = PreLoadBmp2(filename_read2, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  color_depth = GetBmpColorDepth(filename_read2);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);
  byte* ImageData2 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read2, ImgStride, ImgSize, ImageData2, color_depth);

  byte* ImageData3 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * (color_depth / 8) * ImgHeight; i++)
  {
      ImageData3[i] = 0;
  }

  int N = 640 * 480 * (32 / 8);

  /*
  byte* ImageData1;
  byte* ImageData2;

  // Allocate Unified Memory - accessible from CPU or GPU
  hipMallocManaged(&ImageData1, N * sizeof(byte));
  hipMallocManaged(&ImageData2, N * sizeof(byte));

  // initialize ImageData1 and ImageData2 arrays on the host
  for (int i = 0; i < N; i++)
  {
      ImageData1[i] = 3.0f;
      ImageData2[i] = 7.0f;
  }
  */

  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }

  printf("\n");

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  //addKernel << <numBlocks, blockSize >> > (N, ImageData1, ImageData2);
  //addKernel << <1, size >> > (dev_c, dev_a, dev_b);
  //__global__ void addKernel(byte * c, const byte * a, const byte * b)
  
  int size = 480;
  addKernel << <1, size >> > (ImageData3, ImageData1, ImageData2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();


  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write1[] = "ims.new1.bmp";
  char filename_write2[] = "ims.new2.bmp";
  char filename_write3[] = "ims.new3.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write1);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write2);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write3);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  color_depth = 32;
  DumpBmpData(filename_write1, ImageData1, ImgStride, ImgSize, color_depth);
  DumpBmpData(filename_write2, ImageData2, ImgStride, ImgSize, color_depth);
  DumpBmpData(filename_write3, ImageData3, ImgStride, ImgSize, color_depth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP





  /*
  // Free memory
  hipFree(ImageData1);
  hipFree(ImageData2);
  */

  FreePlane(ImageData1);
  FreePlane(ImageData2);
  FreePlane(ImageData3);

  // finalize
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
