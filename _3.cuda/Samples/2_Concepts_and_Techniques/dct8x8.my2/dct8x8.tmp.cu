#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
**************************************************************************
* \file dct8x8.cu
* \brief Contains entry point, wrappers to host and device code and benchmark.
*
* This sample implements forward and inverse Discrete Cosine Transform to blocks
* of image pixels (of 8x8 size), as in JPEG standard. The typical work flow is
*as
* follows:
* 1. Run CPU version (Host code) and measure execution time;
* 2. Run CUDA version (Device code) and measure execution time;
* 3. Output execution timings and calculate CUDA speedup.
*/

#include <stdio.h>
#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

// CUDA kernel to add elements of two arrays
/*
__global__
void addKernel(int n, byte* x, byte* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);

    for (int i = index; i < n; i += stride)
    {
        //�ثe���S���i�ӳo��
        //y[i] = x[i] + y[i];
        y[i] = 0;
        printf(".");
    }
}
*/

__global__ void addKernel(byte* c, const byte* a, const byte* b)
{
    printf("Q");
    int i = threadIdx.x;
    //c[i] = (a[i]/10 + b[i]/10) % 256;
    c[i] = a[i];
    printf("Z");
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const byte* A, const byte* B, byte* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        //C[i] = (A[i] + B[i]) % 256;
        C[i] = (A[i]/2 + B[i]/2) % 256;
    }
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len);

int main(int argc, char **argv)
{
  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  int ImgWidth;
  int ImgHeight;
  int ColorDepth;
  ROI ImgSize;
  int res;
  int ImgStride;
  int ImgDataSize;

  char filename_read1[] = "C:\\______test_files1\\ims01.bmp";
  char filename_read2[] = "C:\\______test_files1\\ims03.bmp";

  printf("Ū���ɮ� : %s\n", filename_read1);
  res = PreLoadBmp2(filename_read1, &ImgWidth, &ImgHeight, &ColorDepth);
  if (res != 0)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  ImgDataSize = ImgWidth * (ColorDepth / 8) * ImgHeight;
  printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
  printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
  byte* ImageData1 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  //printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read1, ImgStride, ImgSize, ImageData1, ColorDepth);

  printf("Ū���ɮ� : %s\n", filename_read2);
  res = PreLoadBmp2(filename_read2, &ImgWidth, &ImgHeight, &ColorDepth);
  if (res != 0)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d\t", ImgSize.width, ImgSize.height);
  printf("�Ϥ��줸�`�� : %d �줸\n", ColorDepth);
  byte* ImageData2 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  //printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read2, ImgStride, ImgSize, ImageData2, ColorDepth);

  byte* ImageData3 = MallocPlaneByte(ImgWidth * (ColorDepth / 8), ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * (ColorDepth / 8) * ImgHeight; i++)
  {
      ImageData3[i] = 0x11;
  }

  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  byte* d_A = NULL;
  err = hipMalloc((void**)&d_A, ImgDataSize);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  byte* d_B = NULL;
  err = hipMalloc((void**)&d_B, ImgDataSize);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  byte* d_C = NULL;
  err = hipMalloc((void**)&d_C, ImgDataSize);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input vectors in device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, ImageData1, ImgDataSize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, ImageData2, ImgDataSize, hipMemcpyHostToDevice);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  int numElements = ImgDataSize/2;  //�����`�j�p

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
  //                   blocks, threads

  err = hipGetLastError();
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");

  //printf("old\n");
  //printData(ImageData1, ImageData2, ImageData3, 20);

  err = hipMemcpy(ImageData3, d_C, ImgDataSize, hipMemcpyDeviceToHost);

  //printf("new\n");
  //printData(ImageData1, ImageData2, ImageData3, 20);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

/*
  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  //addKernel << <numBlocks, blockSize >> > (N, ImageData1, ImageData2);
  //addKernel << <1, size >> > (dev_c, dev_a, dev_b);
  //__global__ void addKernel(byte * c, const byte * a, const byte * b)
  int size = 1;
  addKernel << <1, size >> > (ImageData3, ImageData1, ImageData2);
*/

  // Wait for GPU to finish before accessing on host
  //hipDeviceSynchronize();

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write1[] = "ims.new1.bmp";
  char filename_write2[] = "ims.new2.bmp";
  char filename_write3[] = "ims.new3.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write1);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write2);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write3);
  //ImgStride = 320;

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  ColorDepth = 32;
  DumpBmpData(filename_write1, ImageData1, ImgStride, ImgSize, ColorDepth);
  DumpBmpData(filename_write2, ImageData2, ImgStride, ImgSize, ColorDepth);
  DumpBmpData(filename_write3, ImageData3, ImgStride, ImgSize, ColorDepth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP

  /*
  // Free memory
  hipFree(ImageData1);
  hipFree(ImageData2);
  */

  printf("ImgWidth = %d\tImgHeight=%d\n", ImgWidth, ImgHeight);
  printf("ColorDepth = %d\n", ColorDepth);
  printf("ImgStride = %d\n", ImgStride);
  printf("DataSize = %d\n", ImgWidth * (ColorDepth / 8) * ImgHeight);

  int DataSize = ImgWidth * (ColorDepth / 8) * ImgHeight;

  /*
  byte* data1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte* data2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte* data3 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  */

  byte* data1;
  byte* data2;
  byte* data3;

  hipMalloc((void**)&data1, DataSize);
  hipMalloc((void**)&data2, DataSize);
  hipMalloc((void**)&data3, DataSize);

  hipMemcpy(data1, ImageData1, DataSize, hipMemcpyHostToDevice);
  hipMemcpy(data2, ImageData2, DataSize, hipMemcpyHostToDevice);

  FreePlane(ImageData1);
  FreePlane(ImageData2);
  FreePlane(ImageData3);


  hipFree(data1);
  hipFree(data2);
  hipFree(data3);


  printf("Test PASSED\n");

  err = hipFree(d_A);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


  // finalize
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_A[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_B[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_C[i]);
    }
    printf("\n");
}


