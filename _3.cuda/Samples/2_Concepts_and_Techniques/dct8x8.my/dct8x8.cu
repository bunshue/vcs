#include "hip/hip_runtime.h"
#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

// CUDA kernel to add elements of two arrays
/*
__global__
void addKernel(int n, byte* x, byte* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);

    for (int i = index; i < n; i += stride)
    {
        //�ثe���S���i�ӳo��
        //y[i] = x[i] + y[i];
        y[i] = 0;
        printf(".");
    }
}
*/

__global__ void addKernel(byte* c, const byte* a, const byte* b)
{
    printf("Q");
    int i = threadIdx.x;
    //c[i] = (a[i]/10 + b[i]/10) % 256;
    c[i] = a[i];
    printf("Z");
}


/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/

int main(int argc, char **argv)
{
  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  char filename1[] = "portrait_noise.bmp";
  char filename2[] = "portrait_noise.ok.bmp";

  // preload image (acquire dimensions)
  int ImgWidth;
  int ImgHeight;
  ROI ImgSize;

  printf("PreLoadBmp, file : %s\n", filename1);
  int res = PreLoadBmp(filename1, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  printf("Ū���ɮ� : %s\n", filename1);

  // check image dimensions are multiples of BLOCK_SIZE
  if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0)
  {
    printf("\nError: Input image dimensions must be multiples of 8!\n");
    exit(EXIT_FAILURE);
    return 1;
  }

  // allocate image buffers
  int ImgStride;
  byte *ImgSrc = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDst = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  // load sample image
  LoadBmpAsGray(filename1, ImgStride, ImgSize, ImgSrc);

  printf("�g�J�ɮ� : %s\n", filename2);
  DumpBmpAsGray(filename2, ImgDst, ImgStride, ImgSize);

  //�s�@�@��24�줸�`�פ�bmp�ɮ� ST
  char filename3[] = "my_bmp333b.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename3);
  ImgStride = 320;

  ImgWidth = 16;
  ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  byte* ImgDst333 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * ImgHeight; i++)
  {
      ImgDst333[i] = (i % 256);
  }
  DumpBmpAsGray(filename3, ImgDst333, ImgStride, ImgSize);
  FreePlane(ImgDst333);
  //�s�@�@��24�줸�`�פ�bmp�ɮ� SP

  // release byte planes
  FreePlane(ImgSrc);
  FreePlane(ImgDst);



  //Ū���@��bmp�ɮ� ST, �P�_�줸�`��
  char filename_read[] = "C:\\______test_files\\pic_256X100b.bmp";
  printf("Ū���ɮ� : %s\n", filename_read);

  res = PreLoadBmp2(filename_read, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);

  int color_depth = GetBmpColorDepth(filename_read);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);


  byte* ImageData = MallocPlaneByte(ImgWidth*(color_depth/8), ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  LoadBmpAsData(filename_read, ImgStride, ImgSize, ImageData, color_depth);

  /*
  for (int i = 0; i < 100; i++)
  {
      printf("%02X ", ImageData[i]);


  }
  printf("\n");
  */

  //���Ʀs���t�@��bmp�ɮ�

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write[] = "pic_256X100b.32.new.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  color_depth = 32;
  DumpBmpData(filename_write, ImageData, ImgStride, ImgSize, color_depth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP


  FreePlane(ImageData);



  char filename_read1[] = "C:\\______test_files\\ims01.bmp";
  char filename_read2[] = "C:\\______test_files\\ims03.bmp";

  printf("Ū���ɮ� : %s\n", filename_read1);
  res = PreLoadBmp2(filename_read1, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  color_depth = GetBmpColorDepth(filename_read1);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);
  byte* ImageData1 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read1, ImgStride, ImgSize, ImageData1, color_depth);

  printf("Ū���ɮ� : %s\n", filename_read2);
  res = PreLoadBmp2(filename_read2, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;
  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  color_depth = GetBmpColorDepth(filename_read2);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);
  byte* ImageData2 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  printf("ImgStride = %d\n", ImgStride);
  LoadBmpAsData(filename_read2, ImgStride, ImgSize, ImageData2, color_depth);

  byte* ImageData3 = MallocPlaneByte(ImgWidth * (color_depth / 8), ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * (color_depth / 8) * ImgHeight; i++)
  {
      ImageData3[i] = 0;
  }

  int N = 640 * 480 * (32 / 8);

  /*
  byte* ImageData1;
  byte* ImageData2;

  // Allocate Unified Memory - accessible from CPU or GPU
  hipMallocManaged(&ImageData1, N * sizeof(byte));
  hipMallocManaged(&ImageData2, N * sizeof(byte));

  // initialize ImageData1 and ImageData2 arrays on the host
  for (int i = 0; i < N; i++)
  {
      ImageData1[i] = 3.0f;
      ImageData2[i] = 7.0f;
  }
  */

  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }

  printf("\n");

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  //addKernel << <numBlocks, blockSize >> > (N, ImageData1, ImageData2);
  //addKernel << <1, size >> > (dev_c, dev_a, dev_b);
  //__global__ void addKernel(byte * c, const byte * a, const byte * b)
  
  int size = 480;
  addKernel << <1, size >> > (ImageData3, ImageData1, ImageData2);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();


  for (int i = 0; i < 10; i++)
  {
      printf("ImageData1[%d] = %d\t", i, ImageData1[i]);
      printf("ImageData2[%d] = %d\t", i, ImageData2[i]);
      printf("ImageData3[%d] = %d\n", i, ImageData3[i]);
  }

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write1[] = "ims.new1.bmp";
  char filename_write2[] = "ims.new2.bmp";
  char filename_write3[] = "ims.new3.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write1);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write2);
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write3);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  color_depth = 32;
  DumpBmpData(filename_write1, ImageData1, ImgStride, ImgSize, color_depth);
  DumpBmpData(filename_write2, ImageData2, ImgStride, ImgSize, color_depth);
  DumpBmpData(filename_write3, ImageData3, ImgStride, ImgSize, color_depth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP





  /*
  // Free memory
  hipFree(ImageData1);
  hipFree(ImageData2);
  */

  FreePlane(ImageData1);
  FreePlane(ImageData2);
  FreePlane(ImageData3);

  // finalize
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}


