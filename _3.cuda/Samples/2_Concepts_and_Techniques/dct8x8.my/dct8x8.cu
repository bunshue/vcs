/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
**************************************************************************
* \file dct8x8.cu
* \brief Contains entry point, wrappers to host and device code and benchmark.
*
* This sample implements forward and inverse Discrete Cosine Transform to blocks
* of image pixels (of 8x8 size), as in JPEG standard. The typical work flow is
*as
* follows:
* 1. Run CPU version (Host code) and measure execution time;
* 2. Run CUDA version (Device code) and measure execution time;
* 3. Output execution timings and calculate CUDA speedup.
*/

#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

/**
**************************************************************************
*  Wrapper function for 1st gold version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperGold1(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate float buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);
  float *ImgF2 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // create and start CUDA timer
  StopWatchInterface *timerGold = 0;
  sdkCreateTimer(&timerGold);
  sdkResetTimer(&timerGold);

  // perform block-wise DCT processing and benchmarking
  for (int i = 0; i < BENCHMARK_SIZE; i++) {
    sdkStartTimer(&timerGold);
    computeDCT8x8Gold1(ImgF1, ImgF2, StrideF, Size);
    sdkStopTimer(&timerGold);
  }

  // stop and destroy CUDA timer
  float TimerGoldSpan = sdkGetAverageTimerValue(&timerGold);
  sdkDeleteTimer(&timerGold);

  // perform quantization
  quantizeGoldFloat(ImgF2, StrideF, Size);

  // perform block-wise IDCT processing
  computeIDCT8x8Gold1(ImgF2, ImgF1, StrideF, Size);

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // free float buffers
  FreePlane(ImgF1);
  FreePlane(ImgF2);

  // return time taken by the operation
  return TimerGoldSpan;
}

/**
**************************************************************************
*  Wrapper function for 2nd gold version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperGold2(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate float buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);
  float *ImgF2 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // create and start CUDA timer
  StopWatchInterface *timerGold = 0;
  sdkCreateTimer(&timerGold);
  sdkResetTimer(&timerGold);

  // perform block-wise DCT processing and benchmarking
  for (int i = 0; i < BENCHMARK_SIZE; i++) {
    sdkStartTimer(&timerGold);
    computeDCT8x8Gold2(ImgF1, ImgF2, StrideF, Size);
    sdkStopTimer(&timerGold);
  }

  // stop and destroy CUDA timer
  float TimerGoldSpan = sdkGetAverageTimerValue(&timerGold);
  sdkDeleteTimer(&timerGold);

  // perform quantization
  quantizeGoldFloat(ImgF2, StrideF, Size);

  // perform block-wise IDCT processing
  computeIDCT8x8Gold2(ImgF2, ImgF1, StrideF, Size);

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // free float buffers
  FreePlane(ImgF1);
  FreePlane(ImgF2);

  // return time taken by the operation
  return TimerGoldSpan;
}

/**
**************************************************************************
*  Wrapper function for 1st CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperCUDA1(byte* ImgSrc, byte* ImgDst, int Stride, ROI Size)
{
    // prepare channel format descriptor for passing texture into kernels
    hipChannelFormatDesc floattex = hipCreateChannelDesc<float>();

    // allocate device memory
    hipArray* Src;
    float* Dst;
    size_t DstStride;

    checkCudaErrors(hipMallocArray(&Src, &floattex, Size.width, Size.height));
    checkCudaErrors(hipMallocPitch((void**)(&Dst), &DstStride, Size.width * sizeof(float), Size.height));

    DstStride /= sizeof(float);

    // convert source image to float representation
    int ImgSrcFStride;
    float* ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);

    CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);
    AddFloatPlane(-128.0f, ImgSrcF, ImgSrcFStride, Size);

    // copy from host memory to device
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0, ImgSrcF, ImgSrcFStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    // create and start CUDA timer
    StopWatchInterface* timerCUDA = 0;
    sdkCreateTimer(&timerCUDA);
    sdkResetTimer(&timerCUDA);

    // execute DCT kernel and benchmark
    hipTextureObject_t TexSrc;
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = Src;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&TexSrc, &texRes, &texDescr, NULL));

    for (int i = 0; i < BENCHMARK_SIZE; i++)
    {
        sdkStartTimer(&timerCUDA);
        CUDAkernel1DCT << <grid, threads >> > (Dst, (int)DstStride, 0, 0, TexSrc);
        checkCudaErrors(hipDeviceSynchronize());
        sdkStopTimer(&timerCUDA);
    }

    getLastCudaError("Kernel execution failed");

    // finalize CUDA timer
    float TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);
    sdkDeleteTimer(&timerCUDA);

    // execute Quantization kernel
    CUDAkernelQuantizationFloat << <grid, threads >> > (Dst, (int)DstStride);
    getLastCudaError("Kernel execution failed");

    // copy quantized coefficients from host memory to device array
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0, Dst, DstStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyDeviceToDevice));

    // execute IDCT kernel
    CUDAkernel1IDCT << <grid, threads >> > (Dst, (int)DstStride, 0, 0, TexSrc);
    getLastCudaError("Kernel execution failed");

    // copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(ImgSrcF, ImgSrcFStride * sizeof(float), Dst, DstStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyDeviceToHost));

    // convert image back to byte representation
    AddFloatPlane(128.0f, ImgSrcF, ImgSrcFStride, Size);
    CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

    // clean up memory
    checkCudaErrors(hipDestroyTextureObject(TexSrc));
    checkCudaErrors(hipFreeArray(Src));
    checkCudaErrors(hipFree(Dst));
    FreePlane(ImgSrcF);

    // return time taken by the operation
    return TimerCUDASpan;
}

/**
**************************************************************************
*  Wrapper function for 2nd CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/

float WrapperCUDA2(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate host buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // allocate device memory
  float *src, *dst;
  size_t DeviceStride;
  checkCudaErrors(hipMallocPitch((void **)&src, &DeviceStride,
                                  Size.width * sizeof(float), Size.height));
  checkCudaErrors(hipMallocPitch((void **)&dst, &DeviceStride,
                                  Size.width * sizeof(float), Size.height));
  DeviceStride /= sizeof(float);

  // copy from host memory to device
  checkCudaErrors(hipMemcpy2D(
      src, DeviceStride * sizeof(float), ImgF1, StrideF * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyHostToDevice));

  // create and start CUDA timer
  StopWatchInterface *timerCUDA = 0;
  sdkCreateTimer(&timerCUDA);

  // setup execution parameters
  dim3 GridFullWarps(Size.width / KER2_BLOCK_WIDTH,
                     Size.height / KER2_BLOCK_HEIGHT, 1);
  dim3 ThreadsFullWarps(8, KER2_BLOCK_WIDTH / 8, KER2_BLOCK_HEIGHT / 8);

  // perform block-wise DCT processing and benchmarking
  const int numIterations = 100;

  for (int i = -1; i < numIterations; i++) {
    if (i == 0) {
      checkCudaErrors(hipDeviceSynchronize());
      sdkResetTimer(&timerCUDA);
      sdkStartTimer(&timerCUDA);
    }

    CUDAkernel2DCT<<<GridFullWarps, ThreadsFullWarps>>>(dst, src,
                                                        (int)DeviceStride);
    getLastCudaError("Kernel execution failed");
  }

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timerCUDA);

  // finalize timing of CUDA Kernels
  float avgTime = (float)sdkGetTimerValue(&timerCUDA) / (float)numIterations;
  sdkDeleteTimer(&timerCUDA);
  printf("%f MPix/s //%f ms\n",
         (1E-6 * (float)Size.width * (float)Size.height) / (1E-3 * avgTime),
         avgTime);

  // setup execution parameters for quantization
  dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
  dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

  // execute Quantization kernel
  CUDAkernelQuantizationFloat<<<GridSmallBlocks, ThreadsSmallBlocks>>>(
      dst, (int)DeviceStride);
  getLastCudaError("Kernel execution failed");

  // perform block-wise IDCT processing
  CUDAkernel2IDCT<<<GridFullWarps, ThreadsFullWarps>>>(src, dst,
                                                       (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("Kernel execution failed");

  // copy quantized image block to host
  checkCudaErrors(hipMemcpy2D(
      ImgF1, StrideF * sizeof(float), src, DeviceStride * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyDeviceToHost));

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // clean up memory
  checkCudaErrors(hipFree(dst));
  checkCudaErrors(hipFree(src));
  FreePlane(ImgF1);

  // return time taken by the operation
  return avgTime;
}

/**
**************************************************************************
*  Wrapper function for short CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperCUDAshort(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate host buffers for DCT and other data
  int StrideS;
  short *ImgS1 = MallocPlaneShort(Size.width, Size.height, &StrideS);

  // convert source image to short representation centered at 128
  for (int i = 0; i < Size.height; i++) {
    for (int j = 0; j < Size.width; j++) {
      ImgS1[i * StrideS + j] = (short)ImgSrc[i * Stride + j] - 128;
    }
  }

  // allocate device memory
  short *SrcDst;
  size_t DeviceStride;
  checkCudaErrors(hipMallocPitch((void **)(&SrcDst), &DeviceStride,
                                  Size.width * sizeof(short), Size.height));
  DeviceStride /= sizeof(short);

  // copy from host memory to device
  checkCudaErrors(hipMemcpy2D(
      SrcDst, DeviceStride * sizeof(short), ImgS1, StrideS * sizeof(short),
      Size.width * sizeof(short), Size.height, hipMemcpyHostToDevice));

  // create and start CUDA timer
  StopWatchInterface *timerLibJpeg = 0;
  sdkCreateTimer(&timerLibJpeg);
  sdkResetTimer(&timerLibJpeg);

  // setup execution parameters
  dim3 GridShort(Size.width / KERS_BLOCK_WIDTH, Size.height / KERS_BLOCK_HEIGHT,
                 1);
  dim3 ThreadsShort(8, KERS_BLOCK_WIDTH / 8, KERS_BLOCK_HEIGHT / 8);

  // perform block-wise DCT processing and benchmarking
  sdkStartTimer(&timerLibJpeg);
  CUDAkernelShortDCT<<<GridShort, ThreadsShort>>>(SrcDst, (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timerLibJpeg);
  getLastCudaError("Kernel execution failed");

  // stop and destroy CUDA timer
  float TimerLibJpegSpan16b = sdkGetAverageTimerValue(&timerLibJpeg);
  sdkDeleteTimer(&timerLibJpeg);

  // setup execution parameters for quantization
  dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
  dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

  // execute Quantization kernel
  CUDAkernelQuantizationShort<<<GridSmallBlocks, ThreadsSmallBlocks>>>(
      SrcDst, (int)DeviceStride);
  getLastCudaError("Kernel execution failed");

  // perform block-wise IDCT processing
  CUDAkernelShortIDCT<<<GridShort, ThreadsShort>>>(SrcDst, (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("Kernel execution failed");

  // copy quantized image block to host
  checkCudaErrors(hipMemcpy2D(
      ImgS1, StrideS * sizeof(short), SrcDst, DeviceStride * sizeof(short),
      Size.width * sizeof(short), Size.height, hipMemcpyDeviceToHost));

  // convert image back to byte representation
  for (int i = 0; i < Size.height; i++) {
    for (int j = 0; j < Size.width; j++) {
      ImgDst[i * Stride + j] = clamp_0_255(ImgS1[i * StrideS + j] + 128);
    }
  }

  // free float buffers
  checkCudaErrors(hipFree(SrcDst));
  FreePlane(ImgS1);

  // return time taken by the operation
  return TimerLibJpegSpan16b;
}

/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/

int main(int argc, char **argv)
{
  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  char filename1[] = "portrait_noise.bmp";
  char filename2[] = "portrait_noise.ok.bmp";

  // preload image (acquire dimensions)
  int ImgWidth;
  int ImgHeight;
  ROI ImgSize;

  printf("PreLoadBmp, file : %s\n", filename1);
  int res = PreLoadBmp(filename1, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);
  printf("Ū���ɮ� : %s\n", filename1);

  // check image dimensions are multiples of BLOCK_SIZE
  if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0)
  {
    printf("\nError: Input image dimensions must be multiples of 8!\n");
    exit(EXIT_FAILURE);
    return 1;
  }

  // allocate image buffers
  int ImgStride;
  byte *ImgSrc = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDst = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  // load sample image
  LoadBmpAsGray(filename1, ImgStride, ImgSize, ImgSrc);

  printf("�g�J�ɮ� : %s\n", filename2);
  DumpBmpAsGray(filename2, ImgDst, ImgStride, ImgSize);

  //�s�@�@��24�줸�`�פ�bmp�ɮ� ST
  char filename3[] = "my_bmp333b.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename3);
  ImgStride = 320;

  ImgWidth = 16;
  ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  byte* ImgDst333 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  for (int i = 0; i < ImgWidth * ImgHeight; i++)
  {
      ImgDst333[i] = (i % 256);
  }
  DumpBmpAsGray(filename3, ImgDst333, ImgStride, ImgSize);
  FreePlane(ImgDst333);
  //�s�@�@��24�줸�`�פ�bmp�ɮ� SP

  // release byte planes
  FreePlane(ImgSrc);
  FreePlane(ImgDst);



  //Ū���@��bmp�ɮ� ST, �P�_�줸�`��
  char filename_read[] = "C:\\______test_files\\pic_256X100b.bmp";
  printf("Ū���ɮ� : %s\n", filename_read);

  res = PreLoadBmp2(filename_read, &ImgWidth, &ImgHeight);
  if (res)
  {
      printf("\nError: Image file not found or invalid!\n");
      exit(EXIT_FAILURE);
      return 1;
  }

  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  printf("W = %d, H = %d, BLOCK_SIZE = %d\n", ImgSize.width, ImgSize.height, BLOCK_SIZE);

  int color_depth = GetBmpColorDepth(filename_read);
  printf("�Ϥ��줸�`�� : %d �줸\n", color_depth);


  byte* ImageData = MallocPlaneByte(ImgWidth*(color_depth/8), ImgHeight, &ImgStride);

  printf("ImgStride = %d\n", ImgStride);

  LoadBmpAsData(filename_read, ImgStride, ImgSize, ImageData, color_depth);

  /*
  for (int i = 0; i < 100; i++)
  {
      printf("%02X ", ImageData[i]);


  }
  printf("\n");
  */

  //���Ʀs���t�@��bmp�ɮ�

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� ST
  char filename_write[] = "pic_256X100b.32.new.bmp";
  printf("�s�@�@��bmp�ɮ� : %s\n", filename_write);
  //ImgStride = 320;

  //ImgWidth = 16;
  //ImgHeight = 16;
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  color_depth = 32;
  DumpBmpData(filename_write, ImageData, ImgStride, ImgSize, color_depth);

  //�s�@�@�ӯS�w�줸�`�פ�bmp�ɮ� SP




  FreePlane(ImageData);


  // finalize
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
