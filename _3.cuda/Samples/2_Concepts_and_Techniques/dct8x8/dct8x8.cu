#include "Common.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel_quantization.cuh"

/**
**************************************************************************
*  Wrapper function for 1st CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperCUDA1(byte* ImgSrc, byte* ImgDst, int Stride, ROI Size)
{
    // prepare channel format descriptor for passing texture into kernels
    hipChannelFormatDesc floattex = hipCreateChannelDesc<float>();

    // allocate device memory
    hipArray* Src;
    float* Dst;
    size_t DstStride;
    checkCudaErrors(hipMallocArray(&Src, &floattex, Size.width, Size.height));
    checkCudaErrors(hipMallocPitch((void**)(&Dst), &DstStride, Size.width * sizeof(float), Size.height));
    DstStride /= sizeof(float);

    // convert source image to float representation
    int ImgSrcFStride;
    float* ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);
    CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);
    AddFloatPlane(-128.0f, ImgSrcF, ImgSrcFStride, Size);

    // copy from host memory to device
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0, ImgSrcF, ImgSrcFStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyHostToDevice));

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    // create and start CUDA timer
    StopWatchInterface* timerCUDA = 0;
    sdkCreateTimer(&timerCUDA);
    sdkResetTimer(&timerCUDA);

    // execute DCT kernel and benchmark
    hipTextureObject_t TexSrc;
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = Src;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&TexSrc, &texRes, &texDescr, NULL));

    printf("Size.width = %d\n", Size.width);
    printf("Size.height = %d\n", Size.height);
    for (int i = 0; i < BENCHMARK_SIZE; i++)
    {
        sdkStartTimer(&timerCUDA);
        CUDAkernel1DCT << <grid, threads >> > (Dst, (int)DstStride, 0, 0, TexSrc);
        checkCudaErrors(hipDeviceSynchronize());
        sdkStopTimer(&timerCUDA);
    }

    getLastCudaError("Kernel execution failed");

    // finalize CUDA timer
    float TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);
    sdkDeleteTimer(&timerCUDA);

    // execute Quantization kernel
    CUDAkernelQuantizationFloat << <grid, threads >> > (Dst, (int)DstStride);
    getLastCudaError("Kernel execution failed");

    // copy quantized coefficients from host memory to device array
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0, Dst, DstStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyDeviceToDevice));

    // execute IDCT kernel
    CUDAkernel1IDCT << <grid, threads >> > (Dst, (int)DstStride, 0, 0, TexSrc);
    getLastCudaError("Kernel execution failed");

    // copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(ImgSrcF, ImgSrcFStride * sizeof(float), Dst, DstStride * sizeof(float), Size.width * sizeof(float), Size.height, hipMemcpyDeviceToHost));

    // convert image back to byte representation
    AddFloatPlane(128.0f, ImgSrcF, ImgSrcFStride, Size);
    CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

    // clean up memory
    checkCudaErrors(hipDestroyTextureObject(TexSrc));
    checkCudaErrors(hipFreeArray(Src));
    checkCudaErrors(hipFree(Dst));
    FreePlane(ImgSrcF);

    // return time taken by the operation
    return TimerCUDASpan;
}

/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/

int main(int argc, char** argv)
{
    printf("Starting...\n\n");

    // initialize CUDA
    findCudaDevice(argc, (const char**)argv);

    // source and results image filenames
    char SampleImageFname[] = "teapot512.bmp";
    char SampleImageFnameResCUDA1[] = "teapot512_cuda1.bmp";
    char SampleImageFnameResCUDA2[] = "teapot512_cuda2.bmp";
    char SampleImageFnameResCUDAshort[] = "teapot512_cuda_short.bmp";

    char* pSampleImageFpath = sdkFindFilePath(SampleImageFname, argv[0]);

    if (pSampleImageFpath == NULL)
    {
        printf("dct8x8 could not locate Sample Image <%s>\nExiting...\n", pSampleImageFpath);
        exit(EXIT_FAILURE);
    }

    // preload image (acquire dimensions)
    int ImgWidth, ImgHeight;
    ROI ImgSize;
    int res = PreLoadBmp(pSampleImageFpath, &ImgWidth, &ImgHeight);
    ImgSize.width = ImgWidth;
    ImgSize.height = ImgHeight;

    // CONSOLE INFORMATION: saying hello to user
    printf("CUDA sample DCT/IDCT implementation\n");
    printf("===================================\n");
    printf("Loading test image: %s... ", SampleImageFname);

    if (res)
    {
        printf("\nError: Image file not found or invalid!\n");
        exit(EXIT_FAILURE);
        return 1;
    }

    // check image dimensions are multiples of BLOCK_SIZE
    if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0)
    {
        printf("\nError: Input image dimensions must be multiples of 8!\n");
        exit(EXIT_FAILURE);
        return 1;
    }

    printf("[%d x %d]... ", ImgWidth, ImgHeight);

    // allocate image buffers
    int ImgStride;
    byte* ImgSrc = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
    byte* ImgDstCUDA1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

    // load sample image
    LoadBmpAsGray(pSampleImageFpath, ImgStride, ImgSize, ImgSrc);

    //
    // RUNNING WRAPPERS
    //

    // compute CUDA 1 version of DCT/quantization/IDCT
    printf("Success\nRunning CUDA 1 (GPU) version... ");
    float TimeCUDA1 = WrapperCUDA1(ImgSrc, ImgDstCUDA1, ImgStride, ImgSize);

    //
    // Execution statistics, result saving and validation
    //

    // dump result of CUDA 1 processing
    printf("Success\nDumping result to %s... ", SampleImageFnameResCUDA1);
    DumpBmpAsGray(SampleImageFnameResCUDA1, ImgDstCUDA1, ImgStride, ImgSize);

    // print speed info
    printf("Success\n");

    printf("Processing time (CUDA 1)    : %f ms \n", TimeCUDA1);

    //
    // Finalization
    //

    // release byte planes
    FreePlane(ImgSrc);
    FreePlane(ImgDstCUDA1);

    // finalize
    printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
