#include "hip/hip_runtime.h"
#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include "bicubicTexture_kernel.cuh"

hipArray* d_imageArray = 0;

extern "C" void initTexture(int imageWidth, int imageHeight, uchar * h_data) {
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors(hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight));
    checkCudaErrors(hipMemcpy2DToArray(d_imageArray, 0, 0, h_data, imageWidth * sizeof(uchar), imageWidth * sizeof(uchar), imageHeight, hipMemcpyHostToDevice));
    free(h_data);

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = d_imageArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&texObjLinear, &texRes, &texDescr, NULL));

    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModePoint;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&texObjPoint, &texRes, &texDescr, NULL));
}

extern "C" void freeTexture()
{
    checkCudaErrors(hipDestroyTextureObject(texObjPoint));
    checkCudaErrors(hipDestroyTextureObject(texObjLinear));
    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height, float tx, float ty, float scale, float cx, float cy, dim3 blockSize, dim3 gridSize, int filter_mode, uchar4 * output)
{
    // call CUDA kernel, writing results to PBO memory
    switch (filter_mode)
    {
    case MODE_NEAREST:
        d_render << <gridSize, blockSize >> > (output, width, height, tx, ty, scale, cx, cy, texObjPoint);
        break;

    case MODE_BILINEAR:
        d_render << <gridSize, blockSize >> > (output, width, height, tx, ty, scale, cx, cy, texObjLinear);
        break;

    case MODE_BICUBIC:
        d_renderBicubic << <gridSize, blockSize >> > (output, width, height, tx, ty, scale, cx, cy, texObjPoint);
        break;

    case MODE_FAST_BICUBIC:
        d_renderFastBicubic << <gridSize, blockSize >> > (output, width, height, tx, ty, scale, cx, cy, texObjLinear);
        break;

    case MODE_CATROM:
        d_renderCatRom << <gridSize, blockSize >> > (output, width, height, tx, ty, scale, cx, cy, texObjPoint);
        break;
    }

    getLastCudaError("kernel failed");
}

#endif
