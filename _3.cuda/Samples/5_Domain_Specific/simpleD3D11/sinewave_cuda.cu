#include "hip/hip_runtime.h"
#include <stdio.h>
#include "ShaderStructs.h"
#include "hip/hip_runtime_api.h"
#include "sinewave_cuda.h"

__global__ void sinewave_gen_kernel(Vertex* vertices, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    if (y < height && x < width)
    {
        // write output vertex
        vertices[y * width + x].position.x = u;
        vertices[y * width + x].position.y = w;
        vertices[y * width + x].position.z = v;
        vertices[y * width + x].color.x = 1.0f;
        vertices[y * width + x].color.y = 0.0f;
        vertices[y * width + x].color.z = 0.0f;
        vertices[y * width + x].color.w = 0.0f;
    }
}

Vertex* cudaImportVertexBuffer(void* sharedHandle, hipExternalMemory_t& externalMemory, int meshWidth, int meshHeight)
{
    hipExternalMemoryHandleDesc externalMemoryHandleDesc;
    memset(&externalMemoryHandleDesc, 0, sizeof(externalMemoryHandleDesc));

    externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeD3D11ResourceKmt;
    externalMemoryHandleDesc.size = sizeof(Vertex) * meshHeight * meshWidth;
    externalMemoryHandleDesc.flags = hipExternalMemoryDedicated;
    externalMemoryHandleDesc.handle.win32.handle = sharedHandle;

    checkCudaErrors(hipImportExternalMemory(&externalMemory, &externalMemoryHandleDesc));

    hipExternalMemoryBufferDesc externalMemoryBufferDesc;
    memset(&externalMemoryBufferDesc, 0, sizeof(externalMemoryBufferDesc));
    externalMemoryBufferDesc.offset = 0;
    externalMemoryBufferDesc.size = sizeof(Vertex) * meshHeight * meshWidth;
    externalMemoryBufferDesc.flags = 0;

    Vertex* cudaDevVertptr = NULL;
    checkCudaErrors(hipExternalMemoryGetMappedBuffer((void**)&cudaDevVertptr, externalMemory, &externalMemoryBufferDesc));

    return cudaDevVertptr;
}

void cudaImportKeyedMutex(void* sharedHandle, hipExternalSemaphore_t& extSemaphore)
{
    hipExternalSemaphoreHandleDesc extSemaDesc;
    memset(&extSemaDesc, 0, sizeof(extSemaDesc));
    extSemaDesc.type = cudaExternalSemaphoreHandleTypeKeyedMutexKmt;
    extSemaDesc.handle.win32.handle = sharedHandle;
    extSemaDesc.flags = 0;

    checkCudaErrors(hipImportExternalSemaphore(&extSemaphore, &extSemaDesc));
}

void cudaAcquireSync(hipExternalSemaphore_t& extSemaphore, uint64_t key, unsigned int timeoutMs, hipStream_t streamToRun)
{
    hipExternalSemaphoreWaitParams extSemWaitParams;
    memset(&extSemWaitParams, 0, sizeof(extSemWaitParams));
    extSemWaitParams.params.keyedMutex.key = key;
    extSemWaitParams.params.keyedMutex.timeoutMs = timeoutMs;

    checkCudaErrors(hipWaitExternalSemaphoresAsync(&extSemaphore, &extSemWaitParams, 1, streamToRun));
}

void cudaReleaseSync(hipExternalSemaphore_t& extSemaphore, uint64_t key, hipStream_t streamToRun)
{
    hipExternalSemaphoreSignalParams extSemSigParams;
    memset(&extSemSigParams, 0, sizeof(extSemSigParams));
    extSemSigParams.params.keyedMutex.key = key;

    checkCudaErrors(hipSignalExternalSemaphoresAsync(&extSemaphore, &extSemSigParams, 1, streamToRun));
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void RunSineWaveKernel(hipExternalSemaphore_t& extSemaphore, uint64_t& key, unsigned int timeoutMs, size_t mesh_width, size_t mesh_height, Vertex* cudaDevVertptr, hipStream_t streamToRun)
{
    static float t = 0.0f;
    cudaAcquireSync(extSemaphore, key++, timeoutMs, streamToRun);

    dim3 block(16, 16, 1);
    dim3 grid(mesh_width / 16, mesh_height / 16, 1);
    sinewave_gen_kernel << < grid, block, 0, streamToRun >> > (cudaDevVertptr, mesh_width, mesh_height, t);
    getLastCudaError("sinewave_gen_kernel execution failed.\n");

    cudaReleaseSync(extSemaphore, key, streamToRun);
    t += 0.01f;
}

