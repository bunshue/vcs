#include "hip/hip_runtime.h"
/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

#include "../../../../_6.opengl/_code/Common.h"

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

#define abs(a, b)	(((a) > (b)) ? (a - b) : (b - a))

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 600;
const unsigned int window_height = 600;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;
void* d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x;
int mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0;
float rotate_y = 0.0;
float rotate_x_old = 1.0;
float rotate_y_old = 1.0;
float translate_z = -3.0;

StopWatchInterface* timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
float avgFPS = 0.0f;
unsigned int frameCount = 0;

#define MAX(a,b) ((a > b) ? a : b)

void cleanup();

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// rendering callbacks
void display();
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource);

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    float w = 0;

    float cx = (float)width / 2;
    float cy = (float)height / 2;

    //printf("(%d, %d) ", width, height);
    //printf("(%d, %d) ", x, y);

    float dx = abs(x, cx);
    float dy = abs(y, cy);
    float freq = 20.0f;
    time = 0;

    if ((dx < 80) && (dy < 80))
    {
        //w = cosf(dx * dy / width / height * freq) / 2;
        //w = cosf((1 - u) * (1 - u) + (1 - v) * (1 - v));
    }
    else
    {
        w = 0;
    }
    //w = sqrtf((1 - u) * (1 - u) + (1 - v) * (1 - v));

    //w = 2 - u * u - v * v;
    //w /= 2;
    w = (u + v) / 2;

    //w = sqrtf(dx * dx * dy * dy / width / width / height / height);
    //w = sqrtf(u * u + v * v);

    float alpha = 1.0f;
    if (((x % 5) != 0) && ((y % 5) != 0))
    {
        alpha = 0.0f;
    }

    pos[y * width + x] = make_float4(u, v, w, alpha);

    if ((x == 0) || (y == 0) || (x == (width - 1)) || (y == (height - 1)))
    {
        pos[y * width + x] = make_float4(u, v, 0.0f, 1.0f);
    }

    if ((x == width / 3) || (y == height / 3) || (x == width * 2 / 3) || (y == height * 2 / 3))
    {
        pos[y * width + x] = make_float4(u, v, 0.5f, 1.0f);
    }

    if ((x >= width / 3) && (y >= height / 3) && (x <= width * 2 / 3) && (y <= height * 2 / 3))
    {
        pos[y * width + x] = make_float4(u, v, 1.0f, 1.0f);
    }
}

void launch_kernel(float4* pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
    //printf("w = %d h = %d t = %f, ", mesh_width, mesh_height, time); 256, 256, dt = 0.01

    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //                32                    32             1

    //                   256             256
    //printf("mesh_width = %d mesh_height = %d t = %f, ", mesh_width, mesh_height, time);
    //                8            8            1
    //printf("block.x = %d block.y = %d block.z = %d, ", block.x, block.y, block.z);

    //mesh_width = 256
    //mesh_height = 256
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda GL Interop (VBO): %3.1f fps (Max 100Hz)", avgFPS);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

    glutInitWindowSize(window_width, window_height);    // �]�w�����j�p
    glutInitWindowPosition(1100, 200);  // �]�w������m

    glutCreateWindow("Cuda GL Interop (VBO)");

    glutDisplayFunc(display);       //�]�wcallback function
    glutKeyboardFunc(keyboard0);    //�]�wcallback function
    glutMouseFunc(mouse);           //�]�wcallback function
    glutMotionFunc(motion);         //�]�wcallback function

    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

    glewInit(); // initialize necessary OpenGL extensions

    // default initialization
    //glClearColor(0.0, 0.0, 0.0, 1.0);   //�¦�I��
    glClearColor(1.0, 1.0, 0.0, 1.0);   //����I��

    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();	//�]�m���x�}
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4* dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, *vbo_resource));

    //1048576 bytes
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    //�ϥ�GPU
    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    //�ϥ�CPU
    //TBD
    /*
    //�`�O�L�k��CPU���覡��g���
    int i;
    for (i = 0; i < 10; i++)
    {
        dptr[i] = make_float4(0.3f, 0.5f, 0.7f, 1.0f);
    }
    */

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res, unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{
    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

void display()
{
    //printf("d ");
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    //�e�������
    float color_yellow[4] = { 1.0f, 1.0f, 0.0f, 1.0f };
    draw_boundary(color_yellow, 1.5);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();	//�]�m���x�}
    glTranslatef(0.0, 0.0, translate_z);

    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    /*
    //debug print
    if ((rotate_x_old != rotate_x) && (rotate_y_old != rotate_y))
    {
        printf("rx = %f, ry = %f ", rotate_x, rotate_y);
        rotate_x_old = rotate_x;
        rotate_y_old = rotate_y;
    }
    */

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0); //����
    //glColor3f(0.0, 1.0, 0.0); //���
    //glColor3f(0.0, 0.0, 1.0);   //�Ŧ�
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        //�b�o�̩I�s���s�e��
        //printf("d-");
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1 << button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        //�ƹ�����
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        //�ƹ��k��
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

int main(int argc, char** argv)
{
    printf("Starting...\n");

    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    initGL(&argc, argv);
    findCudaDevice(argc, (const char**)argv);

    glutCloseFunc(cleanup);

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    runCuda(&cuda_vbo_resource);

    glutMainLoop();	//�}�l�D�`��ø�s

    return 0;
}

