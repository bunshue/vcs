#include "hip/hip_runtime.h"
// This example demonstrates how to use the CUDA Direct3D bindings with the runtime API.

// Device code.

#ifndef _SIMPLED3D_KERNEL_CU_
#define _SIMPLED3D_KERNEL_CU_

// includes, C string library
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param pos  pos in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    pos[y * width + x] = make_float4(u, w, v, __int_as_float(0xff00ff00));
}

extern "C" void simpleD3DKernel(float4 * pos, unsigned int width, unsigned int height, float time)
{
    hipError_t error = hipSuccess;

    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    kernel << <grid, block >> > (pos, width, height, time);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("kernel() failed to launch error = %d\n", error);
    }
}

#endif  // #ifndef _SIMPLED3D_KERNEL_CU_
