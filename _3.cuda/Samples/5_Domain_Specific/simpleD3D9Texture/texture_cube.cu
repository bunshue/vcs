#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define PI 3.1415926536f

/*
 * Paint a 2D surface with a moving bulls-eye pattern.  The "face" parameter
 * selects  between 6 different colors to use.  We will use a different color on
 * each face of a  cube map.
 */
__global__ void cuda_kernel_texture_cube(char* surface, int width, int height, size_t pitch, int face, float t)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char* pixel;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't correspond to valid pixels
    if (x >= width || y >= height)
    {
        return;
    }

    // get a pointer to this pixel
    pixel = (unsigned char*)(surface + y * pitch) + 4 * x;

    // populate it
    float theta_x = (2.0f * x) / width - 1.0f;
    float theta_y = (2.0f * y) / height - 1.0f;
    float theta = 2.0f * PI * sqrt(theta_x * theta_x + theta_y * theta_y);
    unsigned char value = 255 * (0.6f + 0.4f * cos(theta + t));

    pixel[3] = 255;  // alpha

    if (face % 2)
    {
        pixel[0] =         // blue
            pixel[1] =     // green
            pixel[2] = 0;  // red
        pixel[face / 2] = value;
    }
    else
    {
        pixel[0] =             // blue
            pixel[1] =         // green
            pixel[2] = value;  // red
        pixel[face / 2] = 0;
    }
}

extern "C" void cuda_texture_cube(void* surface, int width, int height, size_t pitch, int face, float t)
{
    hipError_t error = hipSuccess;

    dim3 Db = dim3(16, 16);  // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

    cuda_kernel_texture_cube << <Dg, Db >> > ((char*)surface, width, height, pitch, face, t);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("cuda_kernel_texture_cube() failed to launch error = %d\n", error);
    }
}
