#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include "hip/hip_runtime_api.h"
#include "helper_multiprocess.h"

static const char shmName[] = "simpleIPCshm";

// For direct NVLINK and PCI-E peers, at max 8 simultaneous peers are allowed
// For NVSWITCH connected peers like DGX-2, simultaneous peers are not limited
// in the same way.
#define MAX_DEVICES (32)
#define DATA_SIZE (64ULL << 20ULL)  // 64MB

#define cpu_atomic_add32(a, x) InterlockedAdd((volatile LONG *)a, x)

typedef struct shmStruct_st
{
    size_t nprocesses;
    int barrier;
    int sense;
    int devices[MAX_DEVICES];
    hipIpcMemHandle_t memHandle[MAX_DEVICES];
    hipIpcEventHandle_t eventHandle[MAX_DEVICES];
} shmStruct;

__global__ void simpleKernel(char* ptr, int sz, char val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (; idx < sz; idx += (gridDim.x * blockDim.x))
    {
        ptr[idx] = val;
    }
}

static void barrierWait(volatile int* barrier, volatile int* sense, unsigned int n)
{
    int count;

    // Check-in
    count = cpu_atomic_add32(barrier, 1);
    if (count == n)  // Last one in
    {
        *sense = 1;
    }
    while (!*sense)
    {
        ;
    }

    // Check-out
    count = cpu_atomic_add32(barrier, -1);
    if (count == 0)  // Last one out
    {
        *sense = 0;
    }
    while (*sense)
    {
        ;
    }
}

static void childProcess(int id)
{
    volatile shmStruct* shm = NULL;
    hipStream_t stream;
    sharedMemoryInfo info;
    size_t procCount, i;
    int blocks = 0;
    int threads = 128;
    hipDeviceProp_t prop;
    std::vector<void*> ptrs;
    std::vector<hipEvent_t> events;
    std::vector<char> verification_buffer(DATA_SIZE);

    if (sharedMemoryOpen(shmName, sizeof(shmStruct), &info) != 0)
    {
        printf("Failed to create shared memory slab\n");
        exit(EXIT_FAILURE);
    }
    shm = (volatile shmStruct*)info.addr;
    procCount = shm->nprocesses;

    printf("Process %d: Starting on device %d...\n", id, shm->devices[id]);

    checkCudaErrors(hipSetDevice(shm->devices[id]));
    checkCudaErrors(hipGetDeviceProperties(&prop, shm->devices[id]));
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks, simpleKernel, threads, 0));
    blocks *= prop.multiProcessorCount;

    // Open and track all the allocations and events created in the master
    // process for use later
    for (i = 0; i < procCount; i++)
    {
        void* ptr = NULL;
        hipEvent_t event;

        // Notice, we don't need to explicitly enable peer access for
        // allocations on other devices.
        checkCudaErrors(hipIpcOpenMemHandle(&ptr, *(hipIpcMemHandle_t*)&shm->memHandle[i], hipIpcMemLazyEnablePeerAccess));
        checkCudaErrors(hipIpcOpenEventHandle(&event, *(hipIpcEventHandle_t*)&shm->eventHandle[i]));

        ptrs.push_back(ptr);
        events.push_back(event);
    }

    // At each iteration of the loop, each sibling process will push work on
    // their respective devices accessing the next peer mapped buffer allocated
    // by the master process (these can come from other sibling processes as
    // well). To coordinate each process' access, we force the stream to wait for
    // the work already accessing this buffer asynchronously through IPC events,
    // allowing the CPU processes to continue to queue more work.
    for (i = 0; i < procCount; i++)
    {
        size_t bufferId = (i + id) % procCount;
        // Wait for the buffer to be accessed to be ready
        checkCudaErrors(hipStreamWaitEvent(stream, events[bufferId], 0));
        // Push a simple kernel on it
        simpleKernel << <blocks, threads, 0, stream >> > ((char*)ptrs[bufferId], DATA_SIZE, id);
        checkCudaErrors(hipGetLastError());
        // Signal that this buffer is ready for the next consumer
        checkCudaErrors(hipEventRecord(events[bufferId], stream));
        // Wait for all my sibling processes to push this stage of their work
        // before proceeding to the next. This prevents siblings from racing
        // ahead and clobbering the recorded event or waiting on the wrong
        // recorded event.
        barrierWait(&shm->barrier, &shm->sense, (unsigned int)procCount);
        if (id == 0)
        {
            printf("Step %lld done\n", (unsigned long long)i);
        }
    }

    // Now wait for my buffer to be ready so I can copy it locally and verify it
    checkCudaErrors(hipStreamWaitEvent(stream, events[id], 0));
    checkCudaErrors(hipMemcpyAsync(&verification_buffer[0], ptrs[id], DATA_SIZE, hipMemcpyDeviceToHost, stream));
    // And wait for all the queued up work to complete
    checkCudaErrors(hipStreamSynchronize(stream));

    printf("Process %d: verifying...\n", id);

    // The contents should have the id of the sibling just after me
    char compareId = (char)((id + 1) % procCount);
    for (unsigned long long j = 0; j < DATA_SIZE; j++)
    {
        if (verification_buffer[j] != compareId)
        {
            printf("Process %d: Verification mismatch at %lld: %d != %d\n", id, j, (int)verification_buffer[j], (int)compareId);
        }
    }

    // Clean up!
    for (i = 0; i < procCount; i++)
    {
        checkCudaErrors(hipIpcCloseMemHandle(ptrs[i]));
        checkCudaErrors(hipEventDestroy(events[i]));
    }

    checkCudaErrors(hipStreamDestroy(stream));
    printf("Process %d complete!\n", id);
}

static void parentProcess(char* app)
{
    sharedMemoryInfo info;
    int devCount, i;
    volatile shmStruct* shm = NULL;
    std::vector<void*> ptrs;
    std::vector<hipEvent_t> events;
    std::vector<Process> processes;

    checkCudaErrors(hipGetDeviceCount(&devCount));

    if (sharedMemoryCreate(shmName, sizeof(*shm), &info) != 0)
    {
        printf("Failed to create shared memory slab\n");
        exit(EXIT_FAILURE);
    }
    shm = (volatile shmStruct*)info.addr;
    memset((void*)shm, 0, sizeof(*shm));

    // Pick all the devices that can access each other's memory for this test
    // Keep in mind that CUDA has minimal support for fork() without a
    // corresponding exec() in the child process, but in this case our
    // spawnProcess will always exec, so no need to worry.
    for (i = 0; i < devCount; i++)
    {
        bool allPeers = true;
        hipDeviceProp_t prop;
        checkCudaErrors(hipGetDeviceProperties(&prop, i));

        // CUDA IPC is only supported on devices with unified addressing
        if (!prop.unifiedAddressing)
        {
            printf("Device %d does not support unified addressing, skipping...\n", i);
            continue;
        }
        // This sample requires two processes accessing each device, so we need
        // to ensure exclusive or prohibited mode is not set
        if (prop.computeMode != hipComputeModeDefault)
        {
            printf("Device %d is in an unsupported compute mode for this sample\n", i);
            continue;
        }

        for (int j = 0; j < shm->nprocesses; j++)
        {
            int canAccessPeerIJ, canAccessPeerJI;
            checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerJI, shm->devices[j], i));
            checkCudaErrors(hipDeviceCanAccessPeer(&canAccessPeerIJ, i, shm->devices[j]));
            if (!canAccessPeerIJ || !canAccessPeerJI)
            {
                allPeers = false;
                break;
            }
        }
        if (allPeers)
        {
            // Enable peers here.  This isn't necessary for IPC, but it will
            // setup the peers for the device.  For systems that only allow 8
            // peers per GPU at a time, this acts to remove devices from CanAccessPeer
            for (int j = 0; j < shm->nprocesses; j++)
            {
                checkCudaErrors(hipSetDevice(i));
                checkCudaErrors(hipDeviceEnablePeerAccess(shm->devices[j], 0));
                checkCudaErrors(hipSetDevice(shm->devices[j]));
                checkCudaErrors(hipDeviceEnablePeerAccess(i, 0));
            }
            shm->devices[shm->nprocesses++] = i;
            if (shm->nprocesses >= MAX_DEVICES)
            {
                break;
            }
        }
        else
        {
            printf("Device %d is not peer capable with some other selected peers, ""skipping\n", i);
        }
    }

    if (shm->nprocesses == 0)
    {
        printf("No CUDA devices support IPC\n");
        exit(EXIT_WAIVED);
    }

    // Now allocate memory and an event for each process and fill the shared
    // memory buffer with the IPC handles to communicate
    for (i = 0; i < shm->nprocesses; i++)
    {
        void* ptr = NULL;
        hipEvent_t event;

        checkCudaErrors(hipSetDevice(shm->devices[i]));
        checkCudaErrors(hipMalloc(&ptr, DATA_SIZE));
        checkCudaErrors(hipIpcGetMemHandle((hipIpcMemHandle_t*)&shm->memHandle[i], ptr));
        checkCudaErrors(hipEventCreate(&event, hipEventDisableTiming | hipEventInterprocess));
        checkCudaErrors(hipIpcGetEventHandle((hipIpcEventHandle_t*)&shm->eventHandle[i], event));

        ptrs.push_back(ptr);
        events.push_back(event);
    }

    // Launch the child processes!
    for (i = 0; i < shm->nprocesses; i++)
    {
        char devIdx[10];
        char* const args[] = { app, devIdx, NULL };
        Process process;

        SPRINTF(devIdx, "%d", i);

        if (spawnProcess(&process, app, args))
        {
            printf("Failed to create process\n");
            exit(EXIT_FAILURE);
        }

        processes.push_back(process);
    }

    // And wait for them to finish
    for (i = 0; i < processes.size(); i++)
    {
        if (waitProcess(&processes[i]) != EXIT_SUCCESS)
        {
            printf("Process %d failed!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Clean up!
    for (i = 0; i < shm->nprocesses; i++)
    {
        checkCudaErrors(hipSetDevice(shm->devices[i]));
        checkCudaErrors(hipEventSynchronize(events[i]));
        checkCudaErrors(hipEventDestroy(events[i]));
        checkCudaErrors(hipFree(ptrs[i]));
    }

    sharedMemoryClose(&info);
}

int main(int argc, char** argv)
{
    printf("\nmain start, argc = %d\n", argc);
    if (argc == 1)
    {
        parentProcess(argv[0]);
    }
    else
    {
        childProcess(atoi(argv[1]));
    }
    return EXIT_SUCCESS;
}

