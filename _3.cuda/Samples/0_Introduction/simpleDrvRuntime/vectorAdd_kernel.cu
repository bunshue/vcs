
#include <hip/hip_runtime.h>
/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Device code
extern "C" __global__ void VecAdd_kernel(const float *A, const float *B,
                                         float *C, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N) C[i] = A[i] + B[i];
}
