#include "hip/hip_runtime.h"
/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

typedef unsigned char byte;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const byte* A, const byte* B, byte* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = (A[i] + B[i]) % 256;
    }
}

_ACRTIMP void __cdecl srand(_In_ unsigned int _Seed);

_Check_return_ _ACRTIMP int __cdecl rand(void);

void printData(byte* h_A, byte* h_B, byte* h_C, int len);

int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(byte);

    printf("[Vector addition of %d elements]\n", numElements);

    byte* h_A = (byte*)malloc(size);
    byte* h_B = (byte*)malloc(size);
    byte* h_C = (byte*)malloc(size);

    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() % 256;
        h_B[i] = rand() % 256;
        h_C[i] = 0;
    }

    byte* d_A = NULL;
    err = hipMalloc((void**)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    byte* d_B = NULL;
    err = hipMalloc((void**)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    byte* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");

    printf("old\n");
    printData(h_A, h_B, h_C, 20);

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("new\n");
    printData(h_A, h_B, h_C, 20);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Test PASSED\n");

    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_A[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_B[i]);
    }
    printf("\n");
    for (int i = 0; i < len; i++)
    {
        printf("%4d", h_C[i]);
    }
    printf("\n");
}
