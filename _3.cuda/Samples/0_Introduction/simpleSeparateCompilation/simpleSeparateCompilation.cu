#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

// STL.
#include <vector>

// CUDA runtime.
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA.
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Device library includes.
#include "simpleDeviceLibrary.cuh"

using std::cout;
using std::endl;

using std::vector;

#define EPS 1e-5

typedef unsigned int uint;
typedef float (*deviceFunc)(float);

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Static device pointers to __device__ functions.
__device__ deviceFunc dMultiplyByTwoPtr = multiplyByTwo;
__device__ deviceFunc dDivideByTwoPtr = divideByTwo;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Transforms vector.
//! Applies the __device__ function "f" to each element of the vector "v".
////////////////////////////////////////////////////////////////////////////////
__global__ void transformVector(float* v, deviceFunc f, uint size)
{
    uint tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        v[tid] = (*f)(v[tid]);
    }
}

int main(int argc, char** argv)
{
    cout << "Starting..." << endl;

    try
    {
        // This will pick the best possible CUDA capable device.
        findCudaDevice(argc, (const char**)argv);

        // Create host vector.
        const uint kVectorSize = 1000;

        vector<float> hVector(kVectorSize);

        for (uint i = 0; i < kVectorSize; ++i)
        {
            hVector[i] = rand() / static_cast<float>(RAND_MAX);
        }

        // Create and populate device vector.
        float* dVector;
        checkCudaErrors(hipMalloc(&dVector, kVectorSize * sizeof(float)));

        checkCudaErrors(hipMemcpy(dVector, &hVector[0], kVectorSize * sizeof(float), hipMemcpyHostToDevice));

        // Kernel configuration, where a one-dimensional
        // grid and one-dimensional blocks are configured.
        const int nThreads = 1024;
        const int nBlocks = 1;

        dim3 dimGrid(nBlocks);
        dim3 dimBlock(nThreads);

        // Test library functions.
        deviceFunc hFunctionPtr;

        hipMemcpyFromSymbol(&hFunctionPtr, HIP_SYMBOL(dMultiplyByTwoPtr), sizeof(deviceFunc));
        transformVector << <dimGrid, dimBlock >> > (dVector, hFunctionPtr, kVectorSize);
        checkCudaErrors(hipGetLastError());

        hipMemcpyFromSymbol(&hFunctionPtr, HIP_SYMBOL(dDivideByTwoPtr), sizeof(deviceFunc));
        transformVector << <dimGrid, dimBlock >> > (dVector, hFunctionPtr, kVectorSize);
        checkCudaErrors(hipGetLastError());

        // Download results.
        vector<float> hResultVector(kVectorSize);

        checkCudaErrors(hipMemcpy(&hResultVector[0], dVector, kVectorSize * sizeof(float), hipMemcpyDeviceToHost));

        // Check results.
        for (int i = 0; i < kVectorSize; ++i)
        {
            if (fabs(hVector[i] - hResultVector[i]) > EPS)
            {
                cout << "Computations were incorrect..." << endl;
                testResult = false;
                break;
            }
        }

        // Free resources.
        if (dVector) checkCudaErrors(hipFree(dVector));
    }
    catch (...)
    {
        cout << "Error occured, exiting..." << endl;
        exit(EXIT_FAILURE);
    }
    cout << "Completed, returned " << (testResult ? "OK" : "ERROR") << endl;

    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

