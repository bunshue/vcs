#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val)
{
    printf("blockIdx.x = %d\n", blockIdx.x);
    printf("blockIdx.y = %d\n", blockIdx.y);
    printf("gridDim.x = %d\n", gridDim.x);
    printf("gridDim.y = %d\n", gridDim.y);
    printf("blockDim.x = %d\n", blockDim.x);
    printf("blockDim.y = %d\n", blockDim.y);
    printf("threadIdx.x = %d\n", threadIdx.x);
    printf("threadIdx.y = %d\n", threadIdx.y);
    printf("threadIdx.z = %d\n", threadIdx.z);

    printf("testKernel [%d, %d]:\tValue is : %d\n",
        blockIdx.y * gridDim.x + blockIdx.x, threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x, val);
}

int main(int argc, char** argv)
{
    /* �S�ƻ�Ϊ�
    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);
    printf("devID = %d\n", devID);

    // Get GPU information
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");
    */

    // Kernel configuration, where a two-dimensional grid and three-dimensional blocks are configured.

    dim3 dimGrid(2, 2);
    dim3 dimBlock(2, 2, 2);

    testKernel << <dimGrid, dimBlock >> > (10);

    hipDeviceSynchronize();

    return EXIT_SUCCESS;
}


