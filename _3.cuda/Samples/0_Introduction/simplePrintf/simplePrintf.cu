#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

//int total_thread = 0;
__global__ void testKernel(int val)
{
    /*
    printf("blockIdx.x = %d\n", blockIdx.x);
    printf("blockIdx.y = %d\n", blockIdx.y);
    printf("gridDim.x = %d\n", gridDim.x);
    printf("gridDim.y = %d\n", gridDim.y);
    printf("blockDim.x = %d\n", blockDim.x);
    printf("blockDim.y = %d\n", blockDim.y);
    printf("threadIdx.x = %d\n", threadIdx.x);
    printf("threadIdx.y = %d\n", threadIdx.y);
    printf("threadIdx.z = %d\n", threadIdx.z);
    */

    printf("testKernel [%d, %d]:\tValue is : %d\n",
        blockIdx.y * gridDim.x + blockIdx.x,
        threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x,
        val);
    //total_thread++;
}

int main(int argc, char** argv)
{
    /* �S�ƻ�Ϊ�
    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);
    printf("devID = %d\n", devID);

    // Get GPU information
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");
    */

    // Kernel configuration, where a two-dimensional grid and three-dimensional blocks are configured.

    dim3 dimGrid(3, 3);     //MXN��block
    dim3 dimBlock(2, 2, 2); //�C��block����AXBXC��thread

    testKernel << <dimGrid, dimBlock >> > (10);

    hipDeviceSynchronize();

    //printf("total_thread  = %d\n", total_thread);

    return EXIT_SUCCESS;
}


