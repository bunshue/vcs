#include "hip/hip_runtime.h"
#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif

// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    assert(gtid < N);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {
    printf("Starting...\n");

    runTest(argc, argv);

    printf("Completed, returned %s\n", testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, char** argv)
{
    int Nblocks = 2;
    int Nthreads = 32;
    hipError_t error;

#ifndef _WIN32
    utsname OS_System_Type;
    uname(&OS_System_Type);

    printf("OS_System_Type.release = %s\n", OS_System_Type.release);

    if (!strcasecmp(OS_System_Type.sysname, "Darwin"))
    {
        printf("simpleAssert is not current supported on Mac OSX\n\n");
        exit(EXIT_SUCCESS);
    }
    else
    {
        printf("OS Info: <%s>\n\n", OS_System_Type.version);
    }

#endif

    // This will pick the best possible CUDA capable device
    findCudaDevice(argc, (const char**)argv);

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    printf("Launch kernel to generate assertion failures\n");
    testKernel << <dimGrid, dimBlock >> > (60);

    // Synchronize (flushes assert output).
    printf("\n-- Begin assert output\n\n");
    error = hipDeviceSynchronize();
    printf("\n-- End assert output\n\n");

    // Check for errors and failed asserts in asynchronous kernel launch.
    if (error == hipErrorAssert)
    {
        printf("Device assert failed as expected, CUDA error message is: %s\n\n", hipGetErrorString(error));
    }

    testResult = error == hipErrorAssert;
}
