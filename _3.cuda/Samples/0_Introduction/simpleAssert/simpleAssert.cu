#include "hip/hip_runtime.h"
#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif

// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    assert(gtid < N);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    printf("Starting...\n");

    int Nblocks = 2;
    int Nthreads = 32;
    hipError_t error;

    // This will pick the best possible CUDA capable device
    findCudaDevice(argc, (const char**)argv);

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    dim3 dimGrid(Nblocks);
    dim3 dimBlock(Nthreads);

    printf("Launch kernel to generate assertion failures\n");
    testKernel << <dimGrid, dimBlock >> > (60);

    // Synchronize (flushes assert output).
    printf("\n-- Begin assert output\n\n");
    error = hipDeviceSynchronize();
    printf("\n-- End assert output\n\n");

    // Check for errors and failed asserts in asynchronous kernel launch.
    if (error == hipErrorAssert)
    {
        printf("Device assert failed as expected, CUDA error message is: %s\n\n", hipGetErrorString(error));
    }

    testResult = error == hipErrorAssert;

    printf("Completed, returned %s\n", testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);

}
