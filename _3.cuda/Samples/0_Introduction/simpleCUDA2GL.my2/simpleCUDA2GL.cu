#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

__global__ void cudaProcess(int* g_odata, int imgw)
{
	
	int tx = threadIdx.x;	//0~15
	int ty = threadIdx.y;	//0~15
	int bw = blockDim.x;	//16
	int bh = blockDim.y;	//16

	//blockIdx.x 0~31
	//blockIdx.y 0~31

	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	//printf("tx = %d, ty = %d, bw = %d, bh = %d, x = %d, y = %d\n", tx, ty, bw, bh, x, y);
	//printf("xx = %d, yy = %d\n", blockIdx.x, blockIdx.y);

	int rr = x % 256;
	int gg = y % 256;
	int bb = 0;

	g_odata[y * imgw + x] = (bb << 16) | (gg << 8) | rr;
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes, int* g_odata, int imgw)
{
	cudaProcess << <grid, block, sbytes >> > (g_odata, imgw);
}


