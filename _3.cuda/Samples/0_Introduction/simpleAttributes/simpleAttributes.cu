#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);

hipAccessPolicyWindow initAccessPolicyWindow(void)
{
    hipAccessPolicyWindow accessPolicyWindow = { 0 };
    accessPolicyWindow.base_ptr = (void*)0;
    accessPolicyWindow.num_bytes = 0;
    accessPolicyWindow.hitRatio = 0.f;
    accessPolicyWindow.hitProp = hipAccessPropertyNormal;
    accessPolicyWindow.missProp = hipAccessPropertyStreaming;
    return accessPolicyWindow;
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param data  input data in global memory
//! @param dataSize  input data size
//! @param bigData  input bigData in global memory
//! @param bigDataSize  input bigData size
//! @param hitcount how many data access are done within block
////////////////////////////////////////////////////////////////////////////////
static __global__ void kernCacheSegmentTest(int* data, int dataSize, int* trash, int bigDataSize, int hitCount)
{
    __shared__ unsigned int hit;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tID = row * blockDim.y + col;
    uint32_t psRand = tID;

    atomicExch(&hit, 0);
    __syncthreads();
    while (hit < hitCount)
    {
        psRand ^= psRand << 13;
        psRand ^= psRand >> 17;
        psRand ^= psRand << 5;

        int idx = tID - psRand;
        if (idx < 0)
        {
            idx = -idx;
        }

        if ((tID % 2) == 0)
        {
            data[psRand % dataSize] = data[psRand % dataSize] + data[idx % dataSize];
        }
        else
        {
            trash[psRand % bigDataSize] = trash[psRand % bigDataSize] + trash[idx % bigDataSize];
        }

        atomicAdd(&hit, 1);
    }
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    bool bTestResult = true;
    hipAccessPolicyWindow accessPolicyWindow;
    hipDeviceProp_t deviceProp;
    hipLaunchAttributeValue streamAttrValue;
    hipStream_t stream;
    hipLaunchAttributeID streamAttrID;
    dim3 threads(32, 32);
    int* dataDevicePointer;
    int* dataHostPointer;
    int dataSize;
    int* bigDataDevicePointer;
    int* bigDataHostPointer;
    int bigDataSize;
    StopWatchInterface* timer = 0;

    printf("Starting... wait... \n\n");

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    // Get device properties
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    dim3 blocks(deviceProp.maxGridSize[1], 1);

    // Make sure device the l2 optimization
    if (deviceProp.persistingL2CacheMaxSize == 0)
    {
        printf("Waiving execution as device %d does not support persisting L2 Caching\n", devID);
        exit(EXIT_WAIVED);
    }

    // Create stream to assiocate with window
    checkCudaErrors(hipStreamCreate(&stream));

    // Set the amount of l2 cache that will be persisting to maximum the device can support
    checkCudaErrors(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, deviceProp.persistingL2CacheMaxSize));

    // Stream attribute to set
    streamAttrID = hipLaunchAttributeAccessPolicyWindow;

    // Default window
    streamAttrValue.accessPolicyWindow = initAccessPolicyWindow();
    accessPolicyWindow = initAccessPolicyWindow();

    // Allocate size of both buffers
    bigDataSize = (deviceProp.l2CacheSize * 4) / sizeof(int);
    dataSize = (deviceProp.l2CacheSize / 4) / sizeof(int);

    // Allocate data
    checkCudaErrors(hipHostMalloc(&dataHostPointer, dataSize * sizeof(int)));
    checkCudaErrors(hipHostMalloc(&bigDataHostPointer, bigDataSize * sizeof(int)));

    for (int i = 0; i < bigDataSize; ++i)
    {
        if (i < dataSize)
        {
            dataHostPointer[i] = i;
        }

        bigDataHostPointer[bigDataSize - i - 1] = i;
    }

    checkCudaErrors(hipMalloc((void**)&dataDevicePointer, dataSize * sizeof(int)));
    checkCudaErrors(hipMalloc((void**)&bigDataDevicePointer, bigDataSize * sizeof(int)));
    checkCudaErrors(hipMemcpyAsync(dataDevicePointer, dataHostPointer, dataSize * sizeof(int), hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(bigDataDevicePointer, bigDataHostPointer, bigDataSize * sizeof(int), hipMemcpyHostToDevice, stream));

    // Make a window for the buffer of interest
    accessPolicyWindow.base_ptr = (void*)dataDevicePointer;
    accessPolicyWindow.num_bytes = dataSize * sizeof(int);
    accessPolicyWindow.hitRatio = 1.f;
    accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
    accessPolicyWindow.missProp = hipAccessPropertyNormal;
    streamAttrValue.accessPolicyWindow = accessPolicyWindow;

    // Assign window to stream
    checkCudaErrors(hipStreamSetAttribute(stream, streamAttrID, &streamAttrValue));

    // Demote any previous persisting lines
    checkCudaErrors(cudaCtxResetPersistingL2Cache());

    checkCudaErrors(hipStreamSynchronize(stream));
    kernCacheSegmentTest << <blocks, threads, 0, stream >> > (dataDevicePointer, dataSize, bigDataDevicePointer, bigDataSize, 0xAFFFF);

    checkCudaErrors(hipStreamSynchronize(stream));
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // Free memory
    checkCudaErrors(hipHostFree(dataHostPointer));
    checkCudaErrors(hipHostFree(bigDataHostPointer));
    checkCudaErrors(hipFree(dataDevicePointer));
    checkCudaErrors(hipFree(bigDataDevicePointer));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

