#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
	r = clamp(r, 0.0f, 255.0f);
	g = clamp(g, 0.0f, 255.0f);
	b = clamp(b, 0.0f, 255.0f);
	return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void cudaProcess(unsigned int* g_odata, int imgw)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;	//0~15
	int ty = threadIdx.y;	//0~15
	int bw = blockDim.x;	//16
	int bh = blockDim.y;	//16

	//blockIdx.x 0~31
	//blockIdx.y 0~31

	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	//printf("tx = %d, ty = %d, bw = %d, bh = %d, x = %d, y = %d\n", tx, ty, bw, bh, x, y);
	//printf("xx = %d, yy = %d\n", blockIdx.x, blockIdx.y);

	unsigned char rr;
	unsigned char gg;
	unsigned char bb;
	unsigned char aa;

	if ((y & 0x20) > 0)
	{
		rr = 100;
	}
	else
	{
		rr = 0;
	}

	gg = 0;

	if ((x & 0x20) > 0)
	{
		bb = 100;
	}
	else
	{
		bb = 0;
	}

	aa = 255;

	//uchar4 c4 = make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0);

	//                      B   G   R   A
	uchar4 c4 = make_uchar4(bb, gg, rr, aa);

	//                                R      G    B
	//g_odata[y * imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
	//g_odata[y * imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
	g_odata[y * imgw + x] = rgbToInt(x % 256, y % 256, 0);

	//g_odata[y * imgw + x] = rgbToInt(0, 255, 0);

	//g_odata[y * imgw + x] = rgbToInt(0, 0, 255);
}

//�}�ѥ�
__global__ void cudaProcess_my(unsigned int* g_odata, int imgw)
{
	
	int tx = threadIdx.x;	//0~15
	int ty = threadIdx.y;	//0~15
	int bw = blockDim.x;	//16
	int bh = blockDim.y;	//16

	//blockIdx.x 0~31
	//blockIdx.y 0~31

	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	//printf("tx = %d, ty = %d, bw = %d, bh = %d, x = %d, y = %d\n", tx, ty, bw, bh, x, y);
	//printf("xx = %d, yy = %d\n", blockIdx.x, blockIdx.y);

	g_odata[y * imgw + x] = rgbToInt(x % 256, y % 256, 0);
}

__global__ void cudaProcess2(unsigned int* g_odata, int imgw)
{
	extern __shared__ uchar4 sdata[];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	unsigned char rr;
	unsigned char gg;
	unsigned char bb;
	unsigned char aa;

	rr = 255;
	gg = 0;
	bb = 0;
	aa = 255;

	//                      B   G   R   A
	uchar4 c4 = make_uchar4(bb, gg, rr, aa);

	int i;
	if (imgw > 200)
	{
		for (i = 0; i < 1048576 / 2; i++)
		{
			//                                R      G    B
			//g_odata[j * imgw + i] = rgbToInt(c4.z, c4.y, c4.x);

			//g_odata[i] = rgbToInt(200, 0, 0);
		}
	}
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes, unsigned int* g_odata, int imgw)
{
	//cudaProcess << <grid, block, sbytes >> > (g_odata, imgw);

	cudaProcess_my << <grid, block, sbytes >> > (g_odata, imgw);

	//cudaProcess2 << <1, 1, 0 >> > (g_odata, imgw);
}

extern "C" void launch_cudaProcess2(unsigned int* g_odata, int imgw)
{
	//cudaProcess2 << <1, 1, 0 >> > (g_odata, imgw);
}



