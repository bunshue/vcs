#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

extern "C" void computeGold(float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(float* g_idata, float* g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern __shared__ float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads();

    // perform some computations
    sdata[tid] = (float)num_threads * sdata[tid];
    __syncthreads();

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

void test_sdkWriteFile()
{
    char* filename = "test.bin";

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;
    float* h_odata = (float*)malloc(mem_size);

    int i;
    for (i = 0; i < num_threads; i++)
    {
        h_odata[i] = (float)i;
    }

    sdkWriteFile(filename, h_odata, num_threads, 0.0f, false);


    free(h_odata);

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    bool bTestResult = true;

    printf("Starting...\n\n");

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    // int devID = findCudaDevice(argc, (const char**)argv);

    StopWatchInterface* timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float* h_idata = (float*)malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float)i;
    }

    // allocate device memory
    float* d_idata;
    checkCudaErrors(hipMalloc((void**)&d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

    // allocate device memory for result
    float* d_odata;
    checkCudaErrors(hipMalloc((void**)&d_odata, mem_size));

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);

    // execute the kernel
    testKernel << <grid, threads, mem_size >> > (d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*)malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads, hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution
    float* reference = (float*)malloc(mem_size);
    computeGold(reference, h_idata, num_threads);

    sdkWriteFile("./dump_data.dat", h_odata, num_threads, 0.0f, false);

    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected solution
    bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    printf("do test_sdkWriteFile\n");

    test_sdkWriteFile();

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}


