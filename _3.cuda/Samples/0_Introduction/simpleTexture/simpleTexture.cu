#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

#define MAX_EPSILON_ERROR 5e-3f

// Define the files that are to be save and the reference images for validation
const char* imageFilename = "teapot512.pgm";
const char* refFilename = "ref_rotated.pgm";

////////////////////////////////////////////////////////////////////////////////
// Constants
const float angle = 0.5f;  // angle to rotate image by (in radians)

// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(float* outputData, int width, int height, float theta, hipTextureObject_t tex)
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = (float)x - (float)width / 2;
    float v = (float)y - (float)height / 2;
    float tu = u * cosf(theta) - v * sinf(theta);
    float tv = v * cosf(theta) + u * sinf(theta);

    tu /= (float)width;
    tv /= (float)height;

    // read from texture and write to global memory
    outputData[y * width + x] = tex2D<float>(tex, tu + 0.5f, tv + 0.5f);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    int devID = findCudaDevice(argc, (const char**)argv);

    // load image from disk
    float* hData = NULL;
    unsigned int width, height;

    printf("imageFilename = %s\n", imageFilename);
    char* imagePath = sdkFindFilePath(imageFilename, argv[0]);

    if (imagePath == NULL)
    {
        printf("Unable to source image file: %s\n", imageFilename);
        exit(EXIT_FAILURE);
    }
    printf("imageFilename : %s\n", imagePath);

    sdkLoadPGM(imagePath, &hData, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

    // Load reference image from image (output)
    float* hDataRef = (float*)malloc(size);

    printf("refFilename = %s\n", refFilename);
    char* refPath = sdkFindFilePath(refFilename, argv[0]);
    printf("refFilename : %s\n", refPath);

    if (refPath == NULL)
    {
        printf("Unable to find reference image file: %s\n", refFilename);
        exit(EXIT_FAILURE);
    }

    sdkLoadPGM(refPath, &hDataRef, &width, &height);

    // Allocate device memory for result
    float* dData = NULL;
    checkCudaErrors(hipMalloc((void**)&dData, size));

    // Allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cuArray;
    checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, hData, size, hipMemcpyHostToDevice));

    hipTextureObject_t tex;
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = cuArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    // Warmup
    transformKernel << <dimGrid, dimBlock, 0 >> > (dData, width, height, angle, tex);

    checkCudaErrors(hipDeviceSynchronize());
    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // Execute the kernel
    transformKernel << <dimGrid, dimBlock, 0 >> > (dData, width, height, angle, tex);

    // Check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    printf("%.2f Mpixels/sec\n", (width * height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
    sdkDeleteTimer(&timer);

    // Allocate mem for the result on host side
    float* hOutputData = (float*)malloc(size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(hOutputData, dData, size, hipMemcpyDeviceToHost));

    // Write result to file
    char outputFilename[1024];
    strcpy(outputFilename, imagePath);
    strcpy(outputFilename + strlen(imagePath) - 4, "_out.pgm");

    sdkSavePGM(outputFilename, hOutputData, width, height);

    printf("outputFilename = %s\n", outputFilename);

    sdkWriteFile<float>("./dump_data.dat", hOutputData, width * height, 0.0f, false);

    // We need to reload the data from disk,
    // because it is inverted upon output
    sdkLoadPGM(outputFilename, &hOutputData, &width, &height);

    printf("Comparing files\n");
    printf("\toutput:    <%s>\n", outputFilename);
    printf("\treference: <%s>\n", refPath);

    testResult = compareData(hOutputData, hDataRef, width * height, MAX_EPSILON_ERROR, 0.15f);

    checkCudaErrors(hipDestroyTextureObject(tex));
    checkCudaErrors(hipFree(dData));
    checkCudaErrors(hipFreeArray(cuArray));
    free(imagePath);
    free(refPath);

    printf("Completed, returned %s\n", testResult ? "OK" : "ERROR!");
    exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

