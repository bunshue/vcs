#include "hip/hip_runtime.h"
/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource* cuda_vbo_resource;
void* d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface* myTimer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
float avgFPS = 0.0f;
unsigned int frameCount = 0;

#define MAX(a,b) ((a > b) ? a : b)

void cleanup();

// GL functionality
bool initGL(int* argc, char** argv);
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res, unsigned int vbo_res_flags);
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);

void myTimerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource** vbo_resource);

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

void launch_kernel(float4* pos, unsigned int mesh_width, unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int* argc, char** argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);

    glutInitWindowSize(window_width, window_height);
    glutInitWindowPosition(1100, 200);

    glutCreateWindow("Cuda GL Interop (VBO)");

    glutDisplayFunc(display);       //�]�wcallback function
    glutKeyboardFunc(keyboard);     //�]�wcallback function
    glutMotionFunc(motion);         //�]�wcallback function

    // initialize necessary OpenGL extensions
    if (!isGLVersionSupported(2, 0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    //glClearColor(0.0, 0.0, 0.0, 1.0);   //�¦�I��
    glClearColor(1.0, 1.0, 0.0, 1.0);   //����I��

    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource** vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    float4* dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

    launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint* vbo, struct hipGraphicsResource** vbo_res, unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint* vbo, struct hipGraphicsResource* vbo_res)
{
    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    //printf("d ");

    // run CUDA kernel to generate vertex positions
    //runCuda(&cuda_vbo_resource);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    //glColor3f(1.0, 0.0, 0.0); //����
    //glColor3f(0.0, 1.0, 0.0); //���
    glColor3f(0.0, 0.0, 1.0);   //�Ŧ�
    glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    g_fAnim += 0.01f;
}

int cnt = 0;
void myTimerEvent(int value)
{
    cnt++;
    printf("%d ", cnt);
    glutTimerFunc(1000, myTimerEvent, 0);
}

void cleanup()
{
    sdkDeleteTimer(&myTimer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    float tt;
    switch (key)
    {
    case 13:
        printf("\n");
        break;

    case 27:
    case 'q':
    case 'Q':
        //���}����
        glutDestroyWindow(glutGetWindow());
        return;

    case '1':
        printf("S ");
        sdkStartTimer(&myTimer);
        break;
    case '2':
        sdkStopTimer(&myTimer);
        tt = sdkGetTimerValue(&myTimer);
        //tt = sdkGetAverageTimerValue(&myTimer);

        sdkResetTimer(&myTimer);

        printf("%d.%03d ��   ", (int)(tt / 1000), ((int)tt) % 1000);
        //printf("%f �@��\n", tt);
        break;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1 << button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    printf("Starting...\n");

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char**)argv);

    initGL(&argc, argv);


    glutDisplayFunc(display);       //�]�wcallback function
    glutKeyboardFunc(keyboard);     //�]�wcallback function
    glutMouseFunc(mouse);           //�]�wcallback function
    glutMotionFunc(motion);         //�]�wcallback function
    glutCloseFunc(cleanup);         //�]�wcallback function

    // timer
    sdkCreateTimer(&myTimer);
    glutTimerFunc(1000, myTimerEvent, 0);

    // create VBO
    createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

    // run the cuda part
    runCuda(&cuda_vbo_resource);

    // start rendering mainloop
    glutMainLoop();

    exit(EXIT_SUCCESS);
}

