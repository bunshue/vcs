﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

#include <windows.h>

void init(void);
void reshape(int w, int h);
void display(void);

void init(void)
{
}

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void display(void)
{
	int i;

	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(-1.0, 11.0, -1.0, 11.0); //窗口坐标范围

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	//画10*10网格
	glColor3f(0.0f, 1.0f, 0.0f); //绿色
	for (i = 0; i <= 10; i++) //11条水平线
	{
		glBegin(GL_LINES);
		glVertex2d(0.0, i * 1.0);
		glVertex2d(10.0, i * 1.0);
		glEnd();
	}
	glBegin(GL_LINES); //11条竖线
	for (i = 0; i <= 10; i++)
	{
		glVertex2d(i * 1.0, 0.0);
		glVertex2d(i * 1.0, 10.0);
	}
	glEnd();

	//在对角线画点
	glColor3f(1.0f, 1.0f, 1.0f); //白色
	glPointSize(10.0f); //点大小
	glBegin(GL_POINTS);
	for (i = 0; i <= 10; i++)
		glVertex2d(i * 1.0, i * 1.0);
	glEnd();
	for (i = 0; i <= 10; i++)
	{
		glBegin(GL_POINTS);
		glVertex2d(i * 1.0, 10.0 - i * 1.0);
		glEnd();
	}

	glFlush();
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowSize(500, 500);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("畫網格");

	init();

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutMainLoop();

	return 0;
}
