﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

//  main.cpp
//  opengl_progress_struct

//#include <GLUT/GLUT.h>
//#include <OpenGL/OpenGL.h>

// 初始化參數
void init() {
    glClearColor(0.1, 0.1, 0.4, 0.0);
    glShadeModel(GL_SMOOTH);
}

// 繪圖回調函數
void display()
{
    printf("d ");
    // 清除之前幀數據
    glClear(GL_COLOR_BUFFER_BIT);

    // 繪制三角形
    glBegin(GL_TRIANGLES);
    glColor3f(1, 0, 0);
    glVertex3f(-1, -1, -5);
    glColor3f(0, 1, 0);
    glVertex3f(1, -1, -5);
    glColor3f(0, 0, 1);
    glVertex3f(0, 1, -5);
    glEnd();
    // 執行繪圖命令
    glFlush();
}

// 窗口大小變化回調函數
void reshape(int w, int h)
{
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)w / (GLfloat)h, 0.1, 100000.0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

int main(int argc, const char* argv[])
{
    // 初始化顯示模式
    glutInit(&argc, const_cast<char**>(argv));
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);

    // 初始化窗口
    glutInitWindowSize(600, 600);
    glutInitWindowPosition(1100, 200);

    glutCreateWindow("Color Map");

    init();

    glutReshapeFunc(reshape);
    glutDisplayFunc(display);

    // 開始主循環繪制
    glutMainLoop();
    return 0;
}

