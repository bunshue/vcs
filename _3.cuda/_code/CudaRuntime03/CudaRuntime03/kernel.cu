#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#define N 256
#include <stdio.h>

__global__ void vecAdd(int* a, int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
int main()
{
    int a[N], b[N], c[N];
    int* dev_a, * dev_b, * dev_c;
    // initialize a and b with real values (NOT SHOWN)
    int size = N * sizeof(int);
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    vecAdd << <1, N >> > (dev_a, dev_b, dev_c);
    //         1 block, N threads


    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
