#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#define N 256
#include <stdio.h>

typedef unsigned char byte;

__global__ void vecAdd(byte* a, byte* b, byte* c)
{
    int i = threadIdx.x;
    //c[i] = a[i] + b[i];
    c[i] = (a[i] + b[i])%256;
}

// Maximum value that can be returned by the rand function:
#define RAND_MAX 0x7fff

_ACRTIMP void __cdecl srand(_In_ unsigned int _Seed);

_Check_return_ _ACRTIMP int __cdecl rand(void);

#if defined _CRT_RAND_S || defined _CRTBLD
_ACRTIMP errno_t __cdecl rand_s(_Out_ unsigned int* _RandomValue);
#endif

int main()
{
    int a[N], b[N], c[N];

    int size = N * sizeof(byte);

    byte* data1;
    byte* data2;
    byte* data3;

    hipMalloc((void**)&data1, size);
    hipMalloc((void**)&data2, size);
    hipMalloc((void**)&data3, size);

    for (int i = 0; i < N; i++)
    {
        a[i] = (i % 256);
        b[i] = (i % 256);
        c[i] = 0x17;
    }

    for (int i = 0; i < 10; i++)
    {
        printf("a[%d] = %d\tb[%d] = %d\tc[%d] = %d\n", i, a[i], i, b[i], i, c[i]);
    }

    hipMemcpy(data1, a, size, hipMemcpyHostToDevice);
    hipMemcpy(data2, b, size, hipMemcpyHostToDevice);

    vecAdd << <1, N >> > (data1, data2, data3);
    //         1 block, N threads

    hipMemcpy(c, data3, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++)
    {
        printf("a[%d] = %d\tb[%d] = %d\tc[%d] = %d\n", i, a[i], i, b[i], i, c[i]);
    }

    hipFree(data1);
    hipFree(data2);
    hipFree(data3);

    //固定種子之random

    srand(200);

    byte data[1000];

    for (int i = 0; i < 1000; i++)
    {
        data[i] = rand() % 256;
    }

    for (int i = 0; i < 1000; i++)
    {
        printf("%d ", data[i]);
        if ((i % 32) == 31)
            printf("\n");
    }
    printf("\n");




    return 0;
}
