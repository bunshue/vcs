﻿//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"


#include <hip/hip_runtime.h>
#include <stdio.h>

typedef unsigned int uint;
typedef unsigned char uchar;

void PrintArray(float* data, int n);
void PrintArray(int* data, int n);
void PrintArray(uchar* data, int n);
void RandomInit(float*, int);
void RandomInit(int*, int);
void init_input(float* a, size_t size);

float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

int main()
{
    printf("測cuda訊息用\n");

    int i;
    int N = 100;

    size_t size = N * sizeof(float);
    float* h_A;
    h_A = (float*)malloc(size);
    PrintArray(h_A, N);
    RandomInit(h_A, N);
    PrintArray(h_A, N);

    size = N * sizeof(int);
    int* h_B;
    h_B = (int*)malloc(size);
    PrintArray(h_B, N);
    RandomInit(h_B, N);
    PrintArray(h_B, N);


    // Free host memory
    if (h_A)
    {
        free(h_A);
    }
    if (h_B)
    {
        free(h_B);
    }



    size = 1 << 24;  // number of elements to reduce

    size_t maxBlocks = 512;

    printf("%zu elements\n", size);

    float* inputVec_h = NULL;

    hipHostMalloc(&inputVec_h, sizeof(float) * size, hipHostMallocDefault);


    init_input(inputVec_h, size);


    printf("david0913: %s:%s(%d) ST\n", __FILE__, __func__, __LINE__);
    printf("david0913: %s:%s(%d) ST\n", __FILE__, __FUNCTION__, __LINE__);

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    printf("有定義\n");
#else
    printf("無定義\n");
#endif

    uchar* h_Data;
    //uint byteCount = 64 * 1048576;
    uint byteCount = 64 * 1024;

    printf("...allocating CPU memory.\n");
    h_Data = (uchar*)malloc(byteCount);

    PrintArray(h_Data, 100);

    printf("...generating input data\n");
    //srand(2009);

    for (i = 0; i < byteCount; i++)
    {
        h_Data[i] = rand() % 256;
    }

    PrintArray(h_Data, 100);

    free(h_Data);


    printf("測試RandFloat\n");
    int DATA_N = 100;
    float* h_AA;

    int DATA_SZ = DATA_N * sizeof(float);

    h_AA = (float*)malloc(DATA_SZ);

    PrintArray(h_AA, 100);

    // Generating input data on CPU
    for (i = 0; i < DATA_N; i++)
    {
        h_AA[i] = RandFloat(0.0f, 1.0f);
    }
    PrintArray(h_AA, 100);

    free(h_AA);




    int random_r;
    int random_g;
    int random_b;

    for (i = 0; i < 100; i++)
    {
        random_r = rand() % 10 - 5;
        random_g = rand() % 10 - 5;
        random_b = rand() % 10 - 5;

        printf("(%d, %d, %d) ", random_r, random_g, random_b);
    }
    printf("\n");


    printf("random 測試 0~1中間的小數\n");
    for (i = 0; i < 10; i++)
    {
        float f;
        f = (float)rand() / (float)RAND_MAX;
        printf("i = %4d\t%g\n", i, f);
    }

    hipError_t cudaStatus;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void PrintArray(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%g ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

void PrintArray(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%d ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

void PrintArray(uchar* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%3d ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = rand() / (float)RAND_MAX;
    }
}

// Allocates an array with random float entries.
void RandomInit(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        //data[i] = rand() / (int)RAND_MAX;
        data[i] = rand() % 10000;
    }
}


void init_input(float* a, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        a[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

