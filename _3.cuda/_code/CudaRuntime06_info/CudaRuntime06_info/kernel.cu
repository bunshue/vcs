﻿//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"


#include <hip/hip_runtime.h>
#include <stdio.h>

void PrintArray(float* data, int n);
void PrintArray(int* data, int n);
void RandomInit(float*, int);
void RandomInit(int*, int);

int main()
{
    printf("測cuda訊息用\n");

    int N = 100;

    size_t size = N * sizeof(float);
    float* h_A;
    h_A = (float*)malloc(size);
    PrintArray(h_A, N);
    RandomInit(h_A, N);
    PrintArray(h_A, N);

    size = N * sizeof(int);
    int* h_B;
    h_B = (int*)malloc(size);
    PrintArray(h_B, N);
    RandomInit(h_B, N);
    PrintArray(h_B, N);


    // Free host memory
    if (h_A)
    {
        free(h_A);
    }
    if (h_B)
    {
        free(h_B);
    }



    hipError_t cudaStatus;




    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void PrintArray(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%g ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

void PrintArray(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%d ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = rand() / (float)RAND_MAX;
    }
}

// Allocates an array with random float entries.
void RandomInit(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        //data[i] = rand() / (int)RAND_MAX;
        data[i] = rand() % 10000;
    }
}
