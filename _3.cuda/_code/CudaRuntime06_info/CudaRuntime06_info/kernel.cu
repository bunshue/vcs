﻿//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"


#include <hip/hip_runtime.h>
#include <stdio.h>

void PrintArray(float* data, int n);
void PrintArray(int* data, int n);
void RandomInit(float*, int);
void RandomInit(int*, int);
void init_input(float* a, size_t size);

int main()
{
    printf("測cuda訊息用\n");

    int N = 100;

    size_t size = N * sizeof(float);
    float* h_A;
    h_A = (float*)malloc(size);
    PrintArray(h_A, N);
    RandomInit(h_A, N);
    PrintArray(h_A, N);

    size = N * sizeof(int);
    int* h_B;
    h_B = (int*)malloc(size);
    PrintArray(h_B, N);
    RandomInit(h_B, N);
    PrintArray(h_B, N);


    // Free host memory
    if (h_A)
    {
        free(h_A);
    }
    if (h_B)
    {
        free(h_B);
    }



    size = 1 << 24;  // number of elements to reduce

    size_t maxBlocks = 512;

    printf("%zu elements\n", size);

    float* inputVec_h = NULL;

    hipHostMalloc(&inputVec_h, sizeof(float) * size, hipHostMallocDefault);


    init_input(inputVec_h, size);


    printf("david0913: %s:%s(%d) ST\n", __FILE__, __func__, __LINE__);
    printf("david0913: %s:%s(%d) ST\n", __FILE__, __FUNCTION__, __LINE__);

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    printf("有定義\n");
#else
    printf("無定義\n");
#endif


    hipError_t cudaStatus;




    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void PrintArray(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%g ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

void PrintArray(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        printf("%d ", data[i]);
        if ((i % 10) == 9)
            printf("\n");
    }
    printf("\n");
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        data[i] = rand() / (float)RAND_MAX;
    }
}

// Allocates an array with random float entries.
void RandomInit(int* data, int n)
{
    for (int i = 0; i < n; i++)
    {
        //data[i] = rand() / (int)RAND_MAX;
        data[i] = rand() % 10000;
    }
}


void init_input(float* a, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        a[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}




