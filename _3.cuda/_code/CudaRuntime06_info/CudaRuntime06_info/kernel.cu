﻿//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"


#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
    printf("測cuda訊息用\n");

    hipError_t cudaStatus;




    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

