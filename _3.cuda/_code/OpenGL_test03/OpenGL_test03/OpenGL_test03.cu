﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

GLubyte rasters[24] = {
   0xc0, 0x00, 0xc0, 0x00, 0xc0, 0x00, 0xc0, 0x00, 0xc0, 0x00,
   0xff, 0x00, 0xff, 0x00, 0xc0, 0x00, 0xc0, 0x00, 0xc0, 0x00,
   0xff, 0xc0, 0xff, 0xc0 };

void init(void)
{
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glClearColor(0.0, 0.0, 0.0, 0.0);
}

/*
void glRasterPos4d(GLdouble x, GLdouble y, GLdouble z = 0, GLdouble w = 1);
void glRasterPos4dv(const GLdouble* v);
//確定當前光柵位置，x,y,z,w指定了當前光柵位置的坐標

glWindowPos(Type x, Type y, Type z);
//用窗口坐標指定當前光柵位置，不必進行矩陣變換、裁剪、或紋理坐標生成。z值被變換為由glDepthRange()設置的當前近側平面值和遠側平面值

void glBitmap(GLsizei, GLsizei height, GLfloat xorig, GLfloat yorig, GLfloat, GLfloat, const GLubyte* bitmap);
//繪制由bitmap指定的位圖，bitmap是一個指向位圖圖像的指針，位圖的原點是當前光柵位置，如果當前光柵位置無效，則這個函數不會繪制任何東西。
//width和height表示位圖的寬度和高度，xorig和yorig定義了位圖的原點，他是根據當期光柵位置確定的，右上為正。
//xmove和ymove表示位圖光柵化之后光柵坐標的x增加值和y增加值
*/

void display(void)
{
    glClear(GL_COLOR_BUFFER_BIT);

    glColor3f(1.0, 0.0, 0.0);
    glRasterPos2i(100, 300);//确定当前光栅位置，x,y,z,w指定了当前光栅位置的坐标
    glBitmap(10, 12, 0.0, 0.0, 11.0, 0.0, rasters);

    glColor3f(0.0, 1.0, 0.0);
    glRasterPos2i(100, 200);//确定当前光栅位置，x,y,z,w指定了当前光栅位置的坐标
    glBitmap(10, 12, 0.0, 0.0, 11.0, 0.0, rasters);

    glColor3f(0.0, 0.0, 1.0);
    glRasterPos2i(100, 100);//确定当前光栅位置，x,y,z,w指定了当前光栅位置的坐标
    glBitmap(10, 12, 0.0, 0.0, 11.0, 0.0, rasters);

    //绘制由bitmap指定的位图，bitmap是一个指向位图图像的指针，位图的原点是当前光栅位置，如果当前光栅位置无效，则这个函数不会绘制任何东西。
    //width和height表示位图的宽度和高度，xorig和yorig定义了位图的原点，他是根据当期光栅位置确定的，右上为正。
    //xmove和ymove表示位图光栅化之后光栅坐标的x增加值和y增加值
    glFlush();
}

void reshape(int w, int h)
{
    glViewport(0, 0, (GLsizei)w, (GLsizei)h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, w, 0, h, -1.0, 1.0);
    glMatrixMode(GL_MODELVIEW);
}

void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
    case 27:
        exit(0);
    }
}

int main(int argc, char** argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);

    glutInitWindowSize(600, 600);
    glutInitWindowPosition(1100, 200);

    glutCreateWindow("測試Bipmap");

    init();

    glutDisplayFunc(display);
    glutReshapeFunc(reshape);
    glutKeyboardFunc(keyboard);

    glutMainLoop();

    return 0;
}

