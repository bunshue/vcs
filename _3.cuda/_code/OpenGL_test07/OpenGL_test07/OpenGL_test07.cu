﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
//#include <GL/glut.h>  //32位元用的

void display(void)
{
    glClear(GL_COLOR_BUFFER_BIT);   //清除窗口
    
    glClear(GL_COLOR_BUFFER_BIT);   // 示例：执行画面清除
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

    //glClearColor(0.1f, 0.2f, 1.f, 1.f); //清除背景 設定顏色

    //glClearColor(1.0, 0.0, 0.0, 1.0);   //清除背景 設定顏色

    glColor4f(1.0, 0.0, 0.0, 1.0);  //設置畫筆顏色為 R
    glRectf(-0.9f, -0.9f, -0.3f, 0.9f);//畫一個矩形

    glColor4f(0.0, 1.0, 0.0, 1.0);  //設置畫筆顏色為 G
    glRectf(-0.4f, -0.8f, 0.4f, 0.8f);//畫一個矩形

    glColor4f(0.0, 0.0, 1.0, 1.0);  //設置畫筆顏色為 B
    glRectf(0.3f, -0.7f, 0.7f, 0.7f);//畫一個矩形


    glFlush();//保證前面的OpenGL命令立即執行   glFlush​​負責刷新繪制緩沖器，保證繪圖命令立即執行。
}

int main(int argc, char* argv[])
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(1100, 200);
    glutInitWindowSize(600, 600);

    glutCreateWindow("第一個OpenGL程序");

    glutDisplayFunc(display);       //設定callback function

    glutMainLoop();

    return 0;
}