#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

//CUDA Runtime API

//下列的範例是以相較於 Driver API 來說比較簡便的 CUDA Runtime API （頁面存檔備份，存於網際網路檔案館） 做列向量的加法：

// 本範例修改自Nvidia官方的CUDA開發指引: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#kernels
// 編譯指令 nvcc vector_add.cu -arch=native -o vector_add.exe
// -arch=native 代表將 device code 編譯成當前電腦 Nvidia GPU 架構的機器碼，拿掉就是照預設編譯成 PTX 中間碼。

#include <stdio.h>
#include <stdlib.h>     //# 引用動態分配 malloc、隨機函數 rand() 和隨機上限 RAND_MAX

typedef unsigned char byte;

#define N 100 // 列向量長度

// Device code: 送入GPU執行的部分

__global__ void VecAdd(byte* A, byte* B, byte* C)
{
	//int kk = blockDim.x * blockIdx.x + threadIdx.x;
	//printf("_%2d%2d%2d%3d\n", blockDim.x, blockIdx.x, threadIdx.x, kk);

	int tid = threadIdx.x; // thread 的 x 座標

	//printf("%4d", tid);

	//C[tid] = (A[tid] + B[tid]) % 256; // 每個 thread 作一次加法	//若使用N個thread, 則每個thread只要做一次加法

	int i;
	for (i = 0; i < 10; i++)
	{
		C[tid*10+i] = (A[tid * 10 + i] + B[tid * 10 + i]) % 256; // 每個 thread 作10次加法	//若使用N/10個thread, 則每個thread要做10次加法
	}

}

// Host code: 送入CPU執行的部分

void printData(byte* h_A, byte* h_B, byte* h_C, int len);

int main()
{
	size_t size = N * sizeof(byte); // 向量的實際大小，以位元組(bytes)為單位

	int i; // 迴圈計數

	// 動態分配位於"host(CPU) 記憶體" 的向量
	byte* h_A = (byte*)malloc(size);
	byte* h_B = (byte*)malloc(size);
	byte* h_C = (byte*)malloc(size);

	// 隨機初始化輸入向量
	for (i = 0; i < N; i++)
	{
		h_A[i] = (byte)(i % 256);
		h_B[i] = (byte)(i % 256);
		h_C[i] = 0;
	}

	printf("size = %d\n", size);
	printf("old\n");
	printData(h_A, h_B, h_C, N);

	// 動態分配位於"device(GPU) 記憶體"的向量
	byte* d_A;
	hipMalloc(&d_A, size); // hipError_t hipMalloc ( void** devPtr, size_t size )
	byte* d_B;
	hipMalloc(&d_B, size);
	byte* d_C;
	hipMalloc(&d_C, size);

	// 將向量從 CPU 複製到 GPU
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	// 將 device code 送入 GPU 並執行，執行時一個 Grid 只有一個 block ，一個 block 有 N 個 thread
	VecAdd << <1, N/10 >> > (d_A, d_B, d_C);	//會執行50次

	// 將算好的向量從 GPU 複製到 CPU
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	// 印出運算結果
	printf("new\n");
	printData(h_A, h_B, h_C, N);

	// 釋放 GPU 記憶體
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// 釋放 CPU 記憶體
	free(h_A);
	free(h_B);
	free(h_C);
}

void printData(byte* h_A, byte* h_B, byte* h_C, int len)
{
	for (int i = 0; i < len; i++)
	{
		printf("%4d", h_A[i]);
	}
	printf("\n");
	for (int i = 0; i < len; i++)
	{
		printf("%4d", h_B[i]);
	}
	printf("\n");
	for (int i = 0; i < len; i++)
	{
		printf("%4d", h_C[i]);
	}
	printf("\n");
}



