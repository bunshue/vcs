﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

GLenum doubleBuffer;
GLint thing1, thing2, thing3, thing4;

static void Init(void)
{
    //           R    G    B     A
    glClearColor(0.0, 0.0, 0.0, 0.0);   //設定背景色(0 0 0為黑色)

    glClearAccum(0.0, 0.0, 0.0, 0.0);

    thing1 = glGenLists(1);
    glNewList(thing1, GL_COMPILE);
    glColor3f(1.0, 0.0, 0.0);   //R
    glRectf(-1.0, -0.8, 1.0, 0.8);
    glEndList();

    thing2 = glGenLists(1);
    glNewList(thing2, GL_COMPILE);
    glColor3f(0.0, 1.0, 0.0);   //G
    glRectf(-0.8, -1.0, 0.2, 1.0);
    glEndList();

    thing3 = glGenLists(1);
    glNewList(thing3, GL_COMPILE);
    glColor3f(0.0, 0.0, 1.0);   //B
    glRectf(-0.2, -1.0, 0.8, 1.0);

    /*
    thing4 = glGenLists(1);
    glNewList(thing4, GL_COMPILE);
    glColor3f(1.0, 0.0, 0.0);   //xxxx
    glRectf(-1.2, -1.2, 1.2, 1.2);
    */

    glEndList();
}

static void reshape(int width, int height)
{
    glViewport(0, 0, width, height);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

static void keyboard(unsigned char key, int x, int y)
{
    switch (key)
    {
    case '1':
        glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
        glutPostRedisplay();
        break;
    case '2':
        glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
        glutPostRedisplay();
        break;
    case 27:
        exit(0);
    }
}

static void display(void)
{
    glPushMatrix();

    glScalef(0.8, 0.8, 1.0);

    glClear(GL_COLOR_BUFFER_BIT);
    glCallList(thing1);
    glAccum(GL_LOAD, 0.5);

    glClear(GL_COLOR_BUFFER_BIT);
    glCallList(thing2);
    glAccum(GL_ACCUM, 0.5);

    glClear(GL_COLOR_BUFFER_BIT);
    glCallList(thing3);
    glAccum(GL_ACCUM, 0.5);

    glClear(GL_COLOR_BUFFER_BIT);
    glCallList(thing4);
    glAccum(GL_ACCUM, 0.5);

    glAccum(GL_RETURN, 1.0);

    glPopMatrix();

    if (doubleBuffer)
    {
        glutSwapBuffers();
    }
    else
    {
        glFlush();
    }
}

static void Args(int argc, char** argv)
{
    GLint i;

    doubleBuffer = GL_FALSE;

    for (i = 1; i < argc; i++)
    {
        if (strcmp(argv[i], "-sb") == 0)
        {
            doubleBuffer = GL_FALSE;
        }
        else if (strcmp(argv[i], "-db") == 0)
        {
            doubleBuffer = GL_TRUE;
        }
    }
}

int main(int argc, char** argv)
{
    GLenum type;

    glutInit(&argc, argv);
    Args(argc, argv);

    type = GLUT_RGB | GLUT_ACCUM;
    type |= (doubleBuffer) ? GLUT_DOUBLE : GLUT_SINGLE;

    glutInitDisplayMode(type);
    glutInitWindowSize(600, 600);

    glutCreateWindow("顏色重疊測試");

    Init();

    glutDisplayFunc(display);       //設定callback function
    glutReshapeFunc(reshape);       //設定callback function
    glutKeyboardFunc(keyboard);     //設定callback function

    glutMainLoop();
}
