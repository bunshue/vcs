#include "hip/hip_runtime.h"
const char *sSDKsample = "simpleStreams";

const char *sEventSyncMethod[] =
{
    "hipEventDefault",
    "hipEventBlockingSync",
    "hipEventDisableTiming",
    NULL
};

const char *sDeviceSyncMethod[] =
{
    "hipDeviceScheduleAuto",
    "hipDeviceScheduleSpin",
    "hipDeviceScheduleYield",
    "INVALID",
    "hipDeviceScheduleBlockingSync",
    NULL
};

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <check.h>

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <stdio.h>
#include <crtdbg.h>
#define UNICODE

//////////////////////////////////////////////////////////////////////
// Nvidia Tools Extension

#define USE_NVTX 1

#if USE_NVTX

#include <nvToolsExt.h>
#include <malloc.h>    // _alloca
#include <stdarg.h>

#define NVTX_ASSERT( COND )    ASSERT_( (COND) )

namespace nvtx
{
    class ScopedRange
    {
    private:
        int m_level;

    public:
        ScopedRange(LPCWSTR wszName)
        {
            m_level = nvtxRangePush(wszName);
        }

        ~ScopedRange()
        {          
            int endLevel = nvtxRangePop();
            //  nvtxRangePop() mismatched level
            NVTX_ASSERT(m_level == endLevel);
        }
    };

    int Print(const char* fmt, ...)
    {
        // printf the string and use the return value of printf to
        // determine the size of the buffer used for vsprintf.
        va_list args;
        va_start(args, fmt);
        int chars = vprintf(fmt, args);

        do
        {
            CheckConditionBreak(chars > 0);

            char* buffer = (char*)_alloca(chars + 1);
            CheckConditionBreak(buffer);

            chars = vsprintf_s(buffer, chars + 1, fmt, args);
            CheckConditionBreak(chars > 0);
            
            nvtxMarkA(buffer);
        } while(0);
        va_end(args);

        return chars;
    }
}

#define NVTX_SCOPED_RANGE( WSTR )   nvtx::ScopedRange nvtxScopedRangeW( (WSTR) )
#define PRINT(FMT, ...)             nvtx::Print(FMT,__VA_ARGS__)

#else	// !USE_NVTX

#define NVTX_SCOPED_RANGE( WSTR )   (0)
#define PRINT(FMT, ...)             printf(FMT,##__VA_ARGS__)

#endif // USE_NVTX


__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=0; i<num_iterations; i++)
    {
        g_data[idx] += *factor;    // non-coalesced on purpose, to burn time
    }
}

int correct_data(int *a, const int n, const int c)
{
    for (int i = 0; i < n; i++)
    {
        if (a[i] != c)
        {
            printf("%d: %d %d\n", i, a[i], c);
            return 0;
        }
    }

    return 1;
}

inline void
AllocateHostMemory(bool bPinGenericMemory, int **pp_a, int **ppAligned_a, int nbytes)
{
#if CUDART_VERSION >= 4000

    if (bPinGenericMemory)
    {
        // allocate a generic page-aligned chunk of system memory
#ifdef WIN32
        printf("> VirtualAlloc() allocating %4.2f Mbytes of (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) VirtualAlloc(NULL, (nbytes + MEMORY_ALIGNMENT), MEM_RESERVE|MEM_COMMIT, PAGE_READWRITE);
#else
        printf("> mmap() allocating %4.2f Mbytes (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) mmap(NULL, (nbytes + MEMORY_ALIGNMENT), PROT_READ|PROT_WRITE, MAP_PRIVATE|MAP_ANON, -1, 0);
#endif

        *ppAligned_a = (int *)ALIGN_UP(*pp_a, MEMORY_ALIGNMENT);

        printf("> hipHostRegister() registering %4.2f Mbytes of generic allocated system memory\n", (float)nbytes/1048576.0f);
        // pin allocate memory
        checkCudaErrors(hipHostRegister(*ppAligned_a, nbytes, hipHostRegisterMapped));
    }
    else
#endif
    {
        printf("> hipHostMalloc() allocating %4.2f Mbytes of system memory\n", (float)nbytes/1048576.0f);
        // allocate host memory (pinned is required for achieve asynchronicity)
        checkCudaErrors(hipHostMalloc((void **)pp_a, nbytes));
        *ppAligned_a = *pp_a;
    }
}

inline void
FreeHostMemory(bool bPinGenericMemory, int **pp_a, int **ppAligned_a, int nbytes)
{
#if CUDART_VERSION >= 4000

    // CUDA 4.0 support pinning of generic host memory
    if (bPinGenericMemory)
    {
        // unpin and delete host memory
        checkCudaErrors(hipHostUnregister(*ppAligned_a));
#ifdef WIN32
        VirtualFree(*pp_a, 0, MEM_RELEASE);
#else
        munmap(*pp_a, nbytes);
#endif
    }
    else
#endif
    {
        hipHostFree(*pp_a);
    }
}

static char *sSyncMethod[] =
{
    "0 (Automatic Blocking)",
    "1 (Spin Blocking)",
    "2 (Yield Blocking)",
    "3 (Undefined Blocking Method)",
    "4 (Blocking Sync Event) = low CPU utilization",
    NULL
};

void printHelp()
{
    printf("Usage: %s [options below]\n", sSDKsample);
    printf("\t--sync_method=n for CPU/GPU synchronization\n");
    printf("\t             n=%s\n", sSyncMethod[0]);
    printf("\t             n=%s\n", sSyncMethod[1]);
    printf("\t             n=%s\n", sSyncMethod[2]);
    printf("\t   <Default> n=%s\n", sSyncMethod[4]);
    printf("\t--use_generic_memory (default) use generic page-aligned for system memory\n");
    printf("\t--use_cuda_malloc_host (optional) use hipHostMalloc to allocate system memory\n");
}

#define DEFAULT_PINNED_GENERIC_MEMORY true

int main(int argc, char **argv)
{
    NVTX_SCOPED_RANGE(L"Main");

    int cuda_device = 0;
    int nstreams = 4;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables
    float scale_factor = 1.0f;

    // allocate generic memory and pin it laster instead of using hipHostAlloc()

    bool bPinGenericMemory  = DEFAULT_PINNED_GENERIC_MEMORY; // we want this to be the default behavior
    int  device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync

    int niterations;    // number of iterations for the loop inside the kernel

    printf("[ %s ]\n\n", sSDKsample);

    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        printHelp();
        return EXIT_SUCCESS;
    }

    if ((device_sync_method = getCmdLineArgumentInt(argc, (const char **)argv, "sync_method")) >= 0)
    {
        if (device_sync_method == 0 || device_sync_method == 1 || device_sync_method == 2 || device_sync_method == 4)
        {
            printf("Device synchronization method set to = %s\n", sSyncMethod[device_sync_method]);
            printf("Setting reps to 100 to demonstrate steady state\n");
            nreps = 100;
        }
        else
        {
            printf("Invalid command line option sync_method=\"%d\"\n", device_sync_method);
            return EXIT_FAILURE;
        }
    }
    else
    {
        printHelp();
        return EXIT_SUCCESS;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "use_generic_memory"))
    {
        bPinGenericMemory = true;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "use_cuda_malloc_host"))
    {
        bPinGenericMemory = false;
    }

    printf("\n> ");
    cuda_device = findCudaDevice(argc, (const char **)argv);

    // check the compute capability of the device
    int num_devices=0;
    checkCudaErrors(hipGetDeviceCount(&num_devices));

    if (0==num_devices)
    {
        printf("your system does not have a CUDA capable device, waiving test...\n");
        return EXIT_WAIVED;
    }

    // check if the command-line chosen device ID is within range, exit if not
    if (cuda_device >= num_devices)
    {
        printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
        return EXIT_FAILURE;
    }

    hipSetDevice(cuda_device);

    // Checking for compute capabilities
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    if ((1 == deviceProp.major) && (deviceProp.minor < 1))
    {
        printf("%s does not have Compute Capability 1.1 or newer.  Reducing workload.\n", deviceProp.name);
    }

    if (deviceProp.major >= 2)
    {
        niterations = 100;
    }
    else
    {
        if (deviceProp.minor > 1)
        {
            niterations = 5;
        }
        else
        {
            niterations = 1; // reduced workload for compute capability 1.0 and 1.1
        }
    }

    // Check if GPU can map host memory (Generic Method), if not then we override bPinGenericMemory to be false
    if (bPinGenericMemory)
    {
        printf("Device: <%s> canMapHostMemory: %s\n", deviceProp.name, deviceProp.canMapHostMemory ? "Yes" : "No");

        if (deviceProp.canMapHostMemory == 0)
        {
            printf("Using hipHostMalloc, CUDA device does not support mapping of generic host memory\n");
            bPinGenericMemory = false;
        }
    }

    // Anything that is less than 32 Cores will have scaled down workload
    scale_factor = max((32.0f / (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * (float)deviceProp.multiProcessorCount)), 1.0f);
    n = (int)rint((float)n / scale_factor);

    printf("> CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);
    printf("> %d Multiprocessor(s) x %d (Cores/Multiprocessor) = %d (Cores)\n",
           deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

    printf("> scale_factor = %1.4f\n", 1.0f/scale_factor);
    printf("> array_size   = %d\n\n", n);

    // enable use of blocking sync, to reduce CPU usage
    printf("> Using CPU/GPU Device Synchronization method (%s)\n", sDeviceSyncMethod[device_sync_method]);
    hipSetDeviceFlags(device_sync_method | (bPinGenericMemory ? hipDeviceMapHost : 0));

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *h_a = 0;                   // pointer to the array data in host memory
    int *hAligned_a = 0;           // pointer to the array data in host memory (aligned to MEMORY_ALIGNMENT)

    // Allocate Host memory (could be using hipHostMalloc or VirtualAlloc/mmap if using the new CUDA 4.0 features
    {
        NVTX_SCOPED_RANGE(L"Init: Host Malloc");
        AllocateHostMemory(bPinGenericMemory, &h_a, &hAligned_a, nbytes);
    }
    

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    {
        NVTX_SCOPED_RANGE(L"Init: Device Malloc");
        checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
        checkCudaErrors(hipMalloc((void **)&d_c, sizeof(int)));
        checkCudaErrors(hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice));
    }

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));

    for (int i = 0; i < nstreams; i++)
    {
        checkCudaErrors(hipStreamCreate(&(streams[i])));
    }

    // create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
    int eventflags = ((device_sync_method == hipDeviceScheduleBlockingSync) ? hipEventBlockingSync: hipEventDefault);

    checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
    checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

    // time memcopy from device
    {
        NVTX_SCOPED_RANGE(L"Time Memcpy DtoH");

        hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
        hipMemcpyAsync(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);   // block until the event is actually recorded
    }

    checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);

    {
        NVTX_SCOPED_RANGE(L"Time Single Kernel");

        hipEventRecord(start_event, 0);

        init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);

        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);
    }

    checkCudaErrors(hipEventElapsedTime(&time_kernel, start_event, stop_event));
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);

    {
        NVTX_SCOPED_RANGE(L"Time Non-Streamed Execution");

        hipEventRecord(start_event, 0);

        for (int k = 0; k < nreps; k++)
        {
            init_array<<<blocks, threads>>>(d_a, d_c, niterations);

            hipMemcpy(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost);
        }
        
        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);
    }

    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(hAligned_a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    
    {
        NVTX_SCOPED_RANGE(L"Time Streamed Execution");

        hipEventRecord(start_event, 0);

        for (int k = 0; k < nreps; k++)
        {
            // asynchronously launch nstreams kernels, each operating on its own portion of data
            for (int i = 0; i < nstreams; i++)
            {
                init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);
            }

            // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
            //   commence executing when all previous CUDA calls in stream x have completed
            for (int i = 0; i < nstreams; i++)
            {
                hipMemcpyAsync(hAligned_a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
            }
        }

        hipEventRecord(stop_event, 0);
        hipEventSynchronize(stop_event);
    }
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    bool bResults = 0 != correct_data(hAligned_a, n, c*nreps*niterations);

    // release resources
    for (int i = 0; i < nstreams; i++)
    {
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // Free hipHostMalloc or Generic Host allocated memory (from CUDA 4.0)
    FreeHostMemory(bPinGenericMemory, &h_a, &hAligned_a, nbytes);

    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();

    return bResults ? EXIT_SUCCESS : EXIT_FAILURE;
}
