﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

#include <windows.h>

void init(void);
void reshape(int w, int h);
void display(void);
void drawGrid(int xmin, int xmax, int ymin, int ymax);

#define NGRID 6

double pnts[][2] = {
	0, 6,
	1, 0,
	2, 6,
	3, 0,
	4, 6,
	5, 0,
	6, 6
};

void init(void)
{
}

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void display(void)
{
	int i, n = 6;

	glClearColor(0.0, 0.0, 0.0, 0.0);
	glClear(GL_COLOR_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0, NGRID, 0.0, NGRID); //窗口坐标范围

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	//画网格
	glColor3f(0.0f, 1.0f, 0.0f); //绿色
	drawGrid(0, NGRID, 0, NGRID);

	//画控制点
	glColor3f(1.0f, 0.0f, 0.0f); //红色
	glPointSize(10.0f); //点大小
	for (i = 0; i <= n; i++)
	{
		glBegin(GL_POINTS);
		glVertex2d(pnts[i][0], pnts[i][1]);
		glEnd();
	}

	//画折线
	glColor3f(1.0f, 1.0f, 1.0f); //白色
	for (i = 0; i < n; i++)
	{
		glBegin(GL_LINES);
		glVertex2d(pnts[i][0], pnts[i][1]);
		glVertex2d(pnts[i + 1][0], pnts[i + 1][1]);
		glEnd();
	}

	glFlush();
}

void drawGrid(int xmin, int xmax, int ymin, int ymax)
{
	int i, j;
	for (j = ymin; j <= ymax; j++) //水平线
	{
		glBegin(GL_LINES);
		glVertex2d(xmin, j);
		glVertex2d(xmax, j);
		glEnd();
	}
	for (i = xmin; i <= xmax; i++) //竖线
	{
		glBegin(GL_LINES);
		glVertex2d(i, ymin);
		glVertex2d(i, ymax);
		glEnd();
	}
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowSize(500, 500);
	glutInitWindowPosition(0, 0);

	glutCreateWindow("畫線範例");
	init();

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutMainLoop();

	return 0;
}

