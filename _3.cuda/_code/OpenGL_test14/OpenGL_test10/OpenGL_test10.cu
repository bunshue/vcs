﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的


#include <windows.h>

#include <stdio.h>

void reshape(int w, int h);
void display(void);

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void display(void)
{
	float mat[16];
	int i;

	glEnable(GL_DEPTH_TEST);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClearDepth(1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(-1.0, 1.0, -1.0, 1.0, -1.0, 1.0);
	glGetFloatv(GL_PROJECTION_MATRIX, mat);
	for (i = 0; i < 16; i++)
	{
		printf("%10.7f", mat[i]);
		if ((i + 1) % 4) printf(" ");
		else printf("\n");
	}

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glColor3f(1.0f, 0.0f, 0.0f); //在右上角画红色平面：应该在后面
	glBegin(GL_POLYGON);
	glVertex3f(0.0f, 0.0f, -1.0f + 0.001f);
	glVertex3f(1.0f, 0.0f, -1.0f + 0.001f);
	glVertex3f(1.0f, 1.0f, -1.0f + 0.001f);
	glVertex3f(0.0f, 1.0f, -1.0f + 0.001f);
	glEnd();
	glColor3f(0.0f, 1.0f, 0.0f); //在左下角画绿色的平面：应该在前面
	glBegin(GL_POLYGON);
	glVertex3f(-1.0f, -1.0f, 1.0f - 0.001f);
	glVertex3f(0.0f + 0.5f, -1.0f, 1.0f - 0.001f);
	glVertex3f(0.0f + 0.5f, 0.0f + 0.5f, 1.0f - 0.001f);
	glVertex3f(-1.0f, 0.0f + 0.5f, 1.0f - 0.001f);
	glEnd();
	glFlush();
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(600, 600);
	glutInitWindowPosition(1100, 200);

	glutCreateWindow("畫顏色色塊");

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutMainLoop();

	return 0;
}

