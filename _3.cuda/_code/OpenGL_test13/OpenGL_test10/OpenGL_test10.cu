#include "hip/hip_runtime.h"
﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <GL/glut.h>  //32位元用的

#include <windows.h>
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>

void init(void);
void reshape(int w, int h);
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void display(void);
void drawCoordinates(void);
void drawTetrahedron(void);

int mx, my; //position of mouse
int m_state = 0; //mouse usage
float x_angle = 0.0f, y_angle = 0.0f; //angle of eye
float dist = 10.0f; //distance from the eye

void init(void)
{
	glEnable(GL_DEPTH_TEST);
}

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
	case '0':
		m_state = 0;
		break;
	case '1':
		m_state = 1;
		break;
	default:
		break;
	}
}

void mouse(int button, int state, int x, int y)
{
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		mx = x;
		my = y;
	}
}

void motion(int x, int y)
{
	GLint dx, dy; //offset of mouse;

	dx = x - mx;
	dy = y - my;

	if (m_state == 0)
	{
		y_angle += dx * 0.1f;
		x_angle += dy * 0.1f;
	}
	else if (m_state == 1)
		dist += (dx + dy) * 0.01f;

	mx = x;
	my = y;

	glutPostRedisplay();
}

void display(void)
{
	double x, y, z, eyex, eyey, eyez;
	int rect[4];
	float w, h;

	glGetIntegerv(GL_VIEWPORT, rect);
	w = rect[2];
	h = rect[3];

	glClearColor(1.0f, 1.0f, 1.0f, 0.0f);
	glClearDepth(1.0);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	if (h < 1) h = 1;
	gluPerspective(30.0, w / h, 0.1, 20.0);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	//glTranslated(0.0, 0.0, -dist);
	//glRotatef(x_angle, 1.0f, 0.0f, 0.0f);
	//glRotatef(y_angle, 0.0f, 1.0f, 0.0f);
	x = 0.0;
	y = 0.0;
	z = dist;
	eyex = x;
	eyey = y * cos(-x_angle * M_PI / 180.0) - z * sin(-x_angle * M_PI / 180.0);
	eyez = y * sin(-x_angle * M_PI / 180.0) + z * cos(-x_angle * M_PI / 180.0);
	x = eyex;
	y = eyey;
	z = eyez;
	eyex = x * cos(-y_angle * M_PI / 180.0) + z * sin(-y_angle * M_PI / 180.0);
	eyey = y;
	eyez = -x * sin(-y_angle * M_PI / 180.0) + z * cos(-y_angle * M_PI / 180.0);
	gluLookAt(eyex, eyey, eyez, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0);
	drawCoordinates();
	glPushMatrix();
	glTranslatef(-1.0f, 0.0f, 0.0f);
	glutWireTeapot(0.5);
	glPopMatrix();
	glPushMatrix();
	glTranslatef(1.0f, 0.0f, 0.0f);
	glScalef(0.5f, 0.5f, 0.5f);
	drawTetrahedron();
	glPopMatrix();

	glFlush();
	glutSwapBuffers();
}

void drawCoordinates(void)
{
	glBegin(GL_LINES);
	glColor3f(1.0f, 0.0f, 0.0f); //画红色的x轴
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(1.0f, 0.0f, 0.0f);
	glColor3f(0.0, 1.0, 0.0); //画绿色的y轴
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(0.0f, 1.0f, 0.0f);
	glColor3f(0.0, 0.0, 1.0); //画蓝色的z轴
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(0.0f, 0.0f, 1.0f);
	glEnd();
}

void drawTetrahedron(void)
{
	float pnt[4][3] = { {0.0,0.0,0.0}, {1.0,0.0,0.0}, {0.0,1.0,0.0}, {0.0,0.0,1.0} };
	int tetra[4][3] = { {0,2,1}, {0,3,2}, {0,1,3}, {1,2,3} };

	glBegin(GL_TRIANGLES);
	glColor3f(1.0f, 0.0f, 0.0f);
	glVertex3fv(pnt[tetra[0][0]]);
	glVertex3fv(pnt[tetra[0][1]]);
	glVertex3fv(pnt[tetra[0][2]]);

	glColor3f(0.0f, 1.0f, 0.0f);
	glVertex3fv(pnt[tetra[1][0]]);
	glVertex3fv(pnt[tetra[1][1]]);
	glVertex3fv(pnt[tetra[1][2]]);

	glColor3f(0.0f, 0.0f, 1.0f);
	glVertex3fv(pnt[tetra[2][0]]);
	glVertex3fv(pnt[tetra[2][1]]);
	glVertex3fv(pnt[tetra[2][2]]);

	glColor3f(0.0f, 1.0f, 1.0f); glVertex3fv(pnt[tetra[3][0]]); //补色
	glColor3f(1.0f, 0.0f, 1.0f); glVertex3fv(pnt[tetra[3][1]]);
	glColor3f(1.0f, 1.0f, 0.0f); glVertex3fv(pnt[tetra[3][2]]);
	glEnd();
}

int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(500, 500);
	glutInitWindowPosition(0, 0);

	glutCreateWindow("畫茶壺");

	init();

	printf("0 keydown means control the angle of the eye\n");
	printf("1 keydown means control the distance of the eye\n");

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);

	glutMainLoop();

	return 0;
}


