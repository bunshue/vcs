﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void init(void);
void reshape(int w, int h);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void display(void);
void drawCoordinates(void);

int mx, my; //position of mouse;
float x_angle, y_angle; //angle of eye

void init(void)
{
}

void reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void mouse(int button, int state, int x, int y)
{
	//MouseDown
	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
	{
		mx = x;
		my = y;
		printf("D(%d, %d) ", mx, my);
	}
}

void motion(int x, int y)
{
	//MouseMove
	int dx, dy; //offset of mouse;

	dx = x - mx;
	dy = y - my;

	y_angle += dx * 0.01f;
	x_angle += dy * 0.01f;

	mx = x;
	my = y;

	printf("M(%d, %d) ", mx, my);

	glutPostRedisplay();
}

void display(void)
{
	printf("d ");
	int rect[4];
	float w, h;

	glGetIntegerv(GL_VIEWPORT, rect);
	w = rect[2];
	h = rect[3];

	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();

	if (w > h)
	{
		glOrtho(-w / h, w / h, -1.0f, 1.0f, -1.0f, 1.0f);
	}
	else
	{
		glOrtho(-1.0f, 1.0f, -h / w, h / w, -1.0f, 1.0f);
	}

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();

	glRotatef(x_angle, 1.0f, 0.0f, 0.0f);
	glRotatef(y_angle, 0.0f, 1.0f, 0.0f);
	drawCoordinates();

	glFlush();
	glutSwapBuffers();
}

void drawCoordinates(void)
{
	glLineWidth(3.0f);

	glColor3f(1.0f, 0.0f, 0.0f); //画红色的x轴
	glBegin(GL_LINES);
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(1.0f, 0.0f, 0.0f);
	glEnd();

	glColor3f(0.0, 1.0, 0.0); //画绿色的y轴
	glBegin(GL_LINES);
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(0.0f, 1.0f, 0.0f);
	glEnd();

	glColor3f(0.0, 0.0, 1.0); //画蓝色的z轴
	glBegin(GL_LINES);
	glVertex3f(0.0f, 0.0f, 0.0f);
	glVertex3f(0.0f, 0.0f, 1.0f);
	glEnd();
}


int main(int argc, char** argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(500, 500);
	glutInitWindowPosition(0, 0);
	glutCreateWindow("gl_1_2");
	init();

	glutDisplayFunc(display);
	glutReshapeFunc(reshape);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutMainLoop();

	return 0;
}


