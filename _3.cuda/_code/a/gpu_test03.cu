
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
    result[i] = a[i] + b[i];
}

void initWith(float num, float *a, int N)
{
	for(int i=0;i<N;i++)
	{
		a[i]=num;
	
	}


}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  initWith(3, a, N);	//??�N?���A?�z�C�N�O�bCPU�U��???��ȡC
  initWith(4, b, N);
  initWith(0, c, N);

  addVectorsInto<<<1, 1>>>(c, a, b, N);
  hipDeviceSynchronize();

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

