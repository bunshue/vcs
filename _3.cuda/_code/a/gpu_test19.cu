
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include <cassert>
struct S {
 int x;
 int *ptr;
 __host__ __device__ S() { }
 __host__ __device__ S(const S &) { ptr = &x; }
};

__global__ void foo(S in) {
 // this assert may fail, because the compiler
 // generated code will memcpy the contents of "in"
 // from host to kernel parameter memory, so the
 // "in.ptr" is not initialized to "&in.x" because
 // the copy constructor is skipped.
 assert(in.ptr == &in.x);
}

int main() {
  S tmp;
  foo<<<1,1>>>(tmp);
  hipDeviceSynchronize();
}


