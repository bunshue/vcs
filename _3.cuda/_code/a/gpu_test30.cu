
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ __managed__ int x, y=2;
__global__  void  kernel() {
    x = 10;
}
int main() {
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStreamAttachMemAsync(stream1, &x);// Associate “x” with stream1.
    hipDeviceSynchronize();              // Wait for “x” attachment to occur.
    kernel<<< 1, 1, 0, stream1 >>>();     // Note: Launches into stream1.
    y = 20;                               // ERROR: “y” is still associated globally 
                                          // with all streams by default
    return  0;
}



