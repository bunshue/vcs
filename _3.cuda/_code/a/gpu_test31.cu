
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


// Managed variable declaration is an extra annotation with __device__
__device__ __managed__  int  x;
__global__  void  kernel() {
    // Reference "x" directly - it's a normal variable on the GPU.
    printf( "GPU sees: x = %d\n" , x);
} 
int  main() {
    // Set "x" from Host code. Note it's just a normal variable on the CPU.
    x = 1234;
 
    // Launch a kernel which uses "x" from the GPU.
    kernel<<< 1, 1 >>>(); 
    hipDeviceSynchronize(); 
    return  0;
}

