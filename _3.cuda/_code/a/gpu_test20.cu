
#include <hip/hip_runtime.h>
#include <cassert>

__managed__ int counter;
struct S1 {
S1() { }
S1(const S1 &) { ++counter; }
};

__global__ void foo(S1) {

/* this assertion may fail, because
   the compiler generates stub
   functions on the host for a kernel
   launch, and they may copy the
   argument by value more than once.
*/
assert(counter == 1);
}

int main() {
S1 V;
foo<<<1,1>>>(V);
hipDeviceSynchronize();
}