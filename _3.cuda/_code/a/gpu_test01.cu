
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;

  printf("index = %d\tn=%d\tstride = %d\n", index, n, stride);
  
  for (int i = index; i < n; i += stride)
  {
      y[i] = x[i] + y[i];
  }
}

int main(void)
{
  int N = 1<<16;
  
  std::cout << "N = " <<N <<std::endl;
  
  float *x, *y;

  // Allocate Unified Memory �V accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 3.0f;
    y[i] = 7.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  //std::cout << "Max error: " << maxError << std::endl;
  
  
  for (int i = 0; i < 10; i++)
  {
    printf("x[%d] = %f\t", i, x[i]);
    printf("y[%d] = %f\n", i, y[i]);
  }
  
  

  // Free memory
  hipFree(x);
  hipFree(y);
  
  std::cout << "OK" << std::endl;
  
  return 0;
}
