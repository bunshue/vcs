
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ __managed__ int x[2];
__device__ __managed__ int y;
__global__ void kernel() {
    x[1] = x[0] + y;
}
int main() {
    x[0] = 3;
    y = 5;
    kernel<<< 1, 1 >>>();
    hipDeviceSynchronize();
    printf("result = %d\n", x[1]); 
    return  0;
}


