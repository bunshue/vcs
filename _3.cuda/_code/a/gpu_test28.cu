
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ __managed__ int x, y=2;
__global__  void  kernel() {
    x = 10;
}
int main() {
    kernel<<< 1, 1 >>>();
    hipDeviceSynchronize();
    y = 20;            //  Success on GPUs not supporing concurrent access
    return  0;
}


