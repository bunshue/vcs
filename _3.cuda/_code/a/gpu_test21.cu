
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

struct S {
 int *ptr;
 S() : ptr(nullptr) { }
 S(const S &) { hipMallocManaged(&ptr, sizeof(int)); }
 ~S() { hipFree(ptr); }
};

__global__ void foo(S in) {
 
  //error: This store may write to memory that has already been
  //       freed (see below).
  *(in.ptr) = 4;
 
}

int main() {
 S V;
 
 /* The object 'V' is first copied by value to a compiler-generated
  * stub function that does the kernel launch, and the stub function
  * bitwise copies the contents of the argument to kernel parameter
  * memory.
  * However, GPU kernel execution is asynchronous with host
  * execution. 
  * As a result, S::~S() will execute when the stub function   returns, releasing allocated memory, even though the kernel may not have finished execution.
  */
 foo<<<1,1>>>(V);
 hipDeviceSynchronize();
}




