
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ __managed__ int ret[1000];
__global__ void AplusB(int a, int b) {
    ret[threadIdx.x] = a + b + threadIdx.x;
}
int main() {
    AplusB<<< 1, 1000 >>>(10, 100);
    hipDeviceSynchronize();
    for(int i = 0; i < 1000; i++)
        printf("%d: A+B = %d\n", i, ret[i]);
    return 0;
}

