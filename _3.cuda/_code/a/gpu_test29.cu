
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ __managed__ int x, y=2;
__global__  void  kernel() {
    x = 10;
}
int main() {
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStreamAttachMemAsync(stream1, &y, 0, hipMemAttachHost);
    hipDeviceSynchronize();          // Wait for Host attachment to occur.
    kernel<<< 1, 1, 0, stream1 >>>(); // Note: Launches into stream1.
    y = 20;                           // Success – a kernel is running but “y” 
                                      // has been associated with no stream.
    return  0;
}

