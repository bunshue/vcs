﻿// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

//#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
//#include <GL/glut.h>  //32位元用的

//下面的示例將演示光照和材質在OpenGL上的應用。

//#include <GLUT/GLUT.h>
//#include <OpenGL/OpenGL.h>


// 初始化參數
void init() {
    GLfloat ambient[] = { 0.0, 0.0, 0.0, 1.0 };
    GLfloat diffuse[] = { 1.0, 1.0, 1.0, 1.0 };
    //    GLfloat specular[] = { 1.0, 1.0, 1.0, 1.0 };
    GLfloat position[] = { 0.0, 0, -1.0, 0.0 };
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LESS);
    glLightfv(GL_LIGHT0, GL_AMBIENT, ambient);
    glLightfv(GL_LIGHT0, GL_DIFFUSE, diffuse);
    //    glLightfv(GL_LIGHT0, GL_SPECULAR, specular);
    glLightfv(GL_LIGHT0, GL_POSITION, position);
    glEnable(GL_LIGHTING);
    glEnable(GL_LIGHT0);
    glClearColor(0.0, 0.1, 0.1, 0.0);
}


// 繪圖回調函數
void display() {
    GLfloat no_mat[] = { 0.0, 0.0, 0.0, 1.0 };
    GLfloat mat_ambient[] = { 0.7, 0.7, 0.7, 1.0 };
    GLfloat mat_ambient_color[] = { 0.8, 0.8, 0.2, 1.0 };
    GLfloat mat_diffuse[] = { 0.1, 0.5, 0.8, 1.0 };
    GLfloat mat_specular[] = { 1.0, 1.0, 1.0, 1.0 };
    GLfloat no_shininess[] = { 0.0 };
    GLfloat low_shininess[] = { 5.0 };
    GLfloat high_shininess[] = { 100.0 };
    GLfloat mat_emission[] = { 0.3, 0.2, 0.2, 0.0 };
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);



    /* 第一行第一列繪制的球僅有漫反射光而無環境光和鏡面光。*/
    glPushMatrix();
    glTranslatef(-3.75, 3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, no_mat);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();



    /* 第一行第二列繪制的球有漫反射光和鏡面光，并有低高光，而無環境光 。*/
    glPushMatrix();
    glTranslatef(-1.25, 3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, no_mat);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);

    glPopMatrix();



    /* 第一行第三列繪制的球有漫反射光和鏡面光，并有很亮的高光，而無環境光 。*/
    glPushMatrix();
    glTranslatef(1.25, 3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, no_mat);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, high_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第一行第四列繪制的球有漫反射光和輻射光，而無環境和鏡面反射光。*/
    glPushMatrix();
    glTranslatef(3.75, 3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, no_mat);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第二行第一列繪制的球有漫反射光和環境光，而鏡面反射光。*/
    glPushMatrix();
    glTranslatef(-3.75, 0.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第二行第二列繪制的球有漫反射光、環境光和鏡面光，且有低高光。*/
    glPushMatrix();
    glTranslatef(-1.25, 0.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第二行第三列繪制的球有漫反射光、環境光和鏡面光，且有很亮的高光。*/
    glPushMatrix();
    glTranslatef(1.25, 0.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, high_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第二行第四列繪制的球有漫反射光、環境光和輻射光，而無鏡面光。*/
    glPushMatrix();
    glTranslatef(3.75, 0.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第三行第一列繪制的球有漫反射光和有顏色的環境光，而無鏡面光。*/
    glPushMatrix();
    glTranslatef(-3.75, -3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient_color);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第三行第二列繪制的球有漫反射光和有顏色的環境光以及鏡面光，且有低高光。*/
    glPushMatrix();
    glTranslatef(-1.25, -3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient_color);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第三行第三列繪制的球有漫反射光和有顏色的環境光以及鏡面光，且有很亮的高光。*/
    glPushMatrix();
    glTranslatef(1.25, -3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient_color);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, high_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, no_mat);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();


    /* 第三行第四列繪制的球有漫反射光和有顏色的環境光以及輻射光，而無鏡面光。*/
    glPushMatrix();
    glTranslatef(3.75, -3.0, 0.0);
    glMaterialfv(GL_FRONT, GL_AMBIENT, mat_ambient_color);
    glMaterialfv(GL_FRONT, GL_DIFFUSE, mat_diffuse);
    glMaterialfv(GL_FRONT, GL_SPECULAR, no_mat);
    glMaterialfv(GL_FRONT, GL_SHININESS, no_shininess);
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glutSolidSphere(1.0, 20, 20);
    glPopMatrix();
    // 執行繪圖命令
    glFlush();
}


// 窗口大小變化回調函數
void reshape(int w, int h) {
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)w / (GLfloat)h, 0.1, 100000.0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(0, 0, 10, 0, 0, -1, 0, 1, 0);
}

int main(int argc, const char* argv[])
{
    // 初始化顯示模式
    glutInit(&argc, const_cast<char**>(argv));
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);

    // 初始化窗口
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(100, 100);
    glutCreateWindow(argv[0]);

    init();

    glutReshapeFunc(reshape);
    glutDisplayFunc(display);

    // 開始主循環繪制
    glutMainLoop();
    return 0;
}
