#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include "imageDenoising.h"

////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
float Max(float x, float y) { return (x > y) ? x : y; }

float Min(float x, float y) { return (x < y) ? x : y; }

int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__device__ float lerpf(float a, float b, float c) { return a + (b - a) * c; }

__device__ float vecLen(float4 a, float4 b)
{
    return ((b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) +
        (b.z - a.z) * (b.z - a.z));
}

__device__ TColor make_color(float r, float g, float b, float a)
{
    return ((int)(a * 255.0f) << 24) | ((int)(b * 255.0f) << 16) | ((int)(g * 255.0f) << 8) | ((int)(r * 255.0f) << 0);
}

////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
// Texture object and channel descriptor for image texture
hipTextureObject_t texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

// CUDA array descriptor
hipArray* a_Src;

////////////////////////////////////////////////////////////////////////////////
// Filtering kernels
////////////////////////////////////////////////////////////////////////////////
#include "imageDenoising_copy_kernel.cuh"

extern "C" hipError_t CUDA_MallocArray(uchar4 * *h_Src, int imageW, int imageH)
{
    hipError_t error;

    error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
    error = hipMemcpy2DToArray(a_Src, 0, 0, *h_Src, sizeof(uchar4) * imageW, sizeof(uchar4) * imageW, imageH, hipMemcpyHostToDevice);

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = a_Src;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = false;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;
    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&texImage, &texRes, &texDescr, NULL));

    return error;
}

extern "C" hipError_t CUDA_FreeArray() { return hipFreeArray(a_Src); }
