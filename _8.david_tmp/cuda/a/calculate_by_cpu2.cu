
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];

  for (int j = index; j < n; j += stride)
  {
      for (int i = index; i < (n-stride); i += stride)
      {
          y[i] = sqrt(x[j] * y[i+stride]);
      }

  }


}

int main(void)
{
  int N = 1<<16;
  
  std::cout << "N = " <<N <<std::endl;
  
  float *x, *y;

  // Allocate Unified Memory �V accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 3.0f;
    y[i] = 7.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  //std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  std::cout << "OK" << std::endl;
  
  return 0;
}
